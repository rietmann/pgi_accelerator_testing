
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(ux[0][0][0][0][1]=(alpha*(u[1][0][0][0][0]+u[-1][0][0][0][0])))
(uy[0][0][0][0][2]=(beta*(u[0][1][0][0][0]+u[0][-1][0][0][0])))
(uz[0][0][0][0][3]=(gamma*(u[0][0][1][0][0]+u[0][0][-1][0][0])))

*/
__global__ void gradient(float *  *  ux_1_0_out, float *  *  uy_2_0_out, float *  *  uz_3_0_out, float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c)
{
/*
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
*/

	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int chunk_idx_x;
	int chunk_idx_x_max;
	int chunk_idx_y;
	int chunk_idx_y_max;
	int chunk_idx_z;
	int chunk_idx_z_max;
	int idx_1_2;
	int size_1_1;
	int size_1_2;
	//int t;
	int thd_idx_x;
	int thd_idx_y;
	int thd_idx_z;
	int thdblks_idx_x;
	int thdblks_idx_x_max;
	int thdblks_idx_y;
	int thdblks_idx_y_max;
	int thdblks_idx_z;
	int thdblks_idx_z_max;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	chunk_idx_x=(c*(threadIdx.x+(blockDim.x*blockIdx.x)));
	chunk_idx_x_max=(chunk_idx_x+c);
	chunk_idx_y=(threadIdx.y+(tmp*blockDim.y));
	chunk_idx_y_max=(chunk_idx_y+1);
	chunk_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	chunk_idx_z_max=(chunk_idx_z+1);
	thdblks_idx_x=(tbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	thdblks_idx_x_max=(thdblks_idx_x+tbx);
	thdblks_idx_y=(tby*(threadIdx.y+(tmp*blockDim.y)));
	thdblks_idx_y_max=(thdblks_idx_y+tby);
	thdblks_idx_z=(tbz*(threadIdx.z+(idx_1_2*blockDim.z)));
	thdblks_idx_z_max=(thdblks_idx_z+tbz);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in thdblks[t=t, s=(tbx, tby, tbz)][0] */
		/* Index bounds calculations for iterators in chunk[t=t, s=(c, 1, 1)][0] */
		/*
		for POINT thd[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in chunk[t=t, s=(:, :, :)][0] parallel 1 <level 2> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
			thd_idx_z=chunk_idx_z;
			thd_idx_y=chunk_idx_y;
			for (thd_idx_x=chunk_idx_x; thd_idx_x<(chunk_idx_x_max-0); thd_idx_x+=1)
			{
				/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
				/*
				u[t=(t+1), s=thd[t=?, s=?][0]][0]=stencil(u[t=t, s=thd[t=?, s=?][0]][0])
				*/
				/* _idx0 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t))+2) */
				_idx0=((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t))+2);
				/* _idx1 = (((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t)) */
				_idx1=(_idx0-2);
				/* _idx2 = ((((thd_idx_z*x_max)*y_max)+(thd_idx_y*x_max))+thd_idx_x) */
				_idx2=((((((_idx1+(((( - x_max)-((2*t)*thd_idx_z))-(2*t))*y_max))+(((((-2*t)*thd_idx_z)-(2*t))-1)*x_max))-((4*(t*t))*thd_idx_z))-((2*t)*thd_idx_y))-(4*(t*t)))-(2*t));
				ux_1_0[_idx2]=(alpha*(u_0_0[_idx0]+u_0_0[_idx1]));
				/* _idx3 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+2)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(4*t))+1) */
				_idx3=(((_idx1+x_max)+(2*t))+1);
				/* _idx4 = (((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+1) */
				_idx4=(((_idx1-x_max)-(2*t))+1);
				uy_2_0[_idx2]=(beta*(u_0_0[_idx3]+u_0_0[_idx4]));
				/* _idx5 = ((((((((((((thd_idx_z+2)*x_max)+((2*t)*thd_idx_z))+(4*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(4*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(8*(t*t)))+(2*t))+1) */
				_idx5=((((_idx1+((x_max+(2*t))*y_max))+((2*t)*x_max))+(4*(t*t)))+1);
				/* _idx6 = (((((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(2*t))+1) */
				_idx6=((((_idx1+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
				uz_3_0[_idx2]=(gamma*(u_0_0[_idx5]+u_0_0[_idx6]));
			}
		}
	}
}

__global__ void initialize(float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c)
{
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int chunk_idx_x;
	int chunk_idx_x_max;
	int chunk_idx_y;
	int chunk_idx_y_max;
	int chunk_idx_z;
	int chunk_idx_z_max;
	int idx_1_2;
	int size_1_1;
	int size_1_2;
	//int t;
	int thd_idx_x;
	int thd_idx_y;
	int thd_idx_z;
	int thdblks_idx_x;
	int thdblks_idx_x_max;
	int thdblks_idx_y;
	int thdblks_idx_y_max;
	int thdblks_idx_z;
	int thdblks_idx_z_max;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	chunk_idx_x=(c*(threadIdx.x+(blockDim.x*blockIdx.x)));
	chunk_idx_x_max=(chunk_idx_x+c);
	chunk_idx_y=(threadIdx.y+(tmp*blockDim.y));
	chunk_idx_y_max=(chunk_idx_y+1);
	chunk_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	chunk_idx_z_max=(chunk_idx_z+1);
	thdblks_idx_x=(tbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	thdblks_idx_x_max=(thdblks_idx_x+tbx);
	thdblks_idx_y=(tby*(threadIdx.y+(tmp*blockDim.y)));
	thdblks_idx_y_max=(thdblks_idx_y+tby);
	thdblks_idx_z=(tbz*(threadIdx.z+(idx_1_2*blockDim.z)));
	thdblks_idx_z_max=(thdblks_idx_z+tbz);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in thdblks[t=t, s=(tbx, tby, tbz)][0] */
		/* Index bounds calculations for iterators in chunk[t=t, s=(c, 1, 1)][0] */
		/*
		for POINT thd[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in chunk[t=t, s=(:, :, :)][0] parallel 1 <level 2> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
			thd_idx_z=chunk_idx_z;
			thd_idx_y=chunk_idx_y;
			for (thd_idx_x=chunk_idx_x; thd_idx_x<(chunk_idx_x_max-0); thd_idx_x+=1)
			{
				/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
				/*
				u[t=(t+1), s=thd[t=?, s=?][0]][0]=stencil(u[t=t, s=thd[t=?, s=?][0]][0])
				*/
				/* _idx0 = (((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t)) */
				_idx0=(((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t));
				u_0_0[_idx0]=0.1;
				/* _idx1 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t))+2) */
				_idx1=(_idx0+2);
				u_0_0[_idx1]=0.1;
				/* _idx2 = ((((thd_idx_z*x_max)*y_max)+(thd_idx_y*x_max))+thd_idx_x) */
				_idx2=((((((_idx0+(((( - x_max)-((2*t)*thd_idx_z))-(2*t))*y_max))+(((((-2*t)*thd_idx_z)-(2*t))-1)*x_max))-((4*(t*t))*thd_idx_z))-((2*t)*thd_idx_y))-(4*(t*t)))-(2*t));
				ux_1_0[_idx2]=0.2;
				/* _idx3 = (((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+1) */
				_idx3=(((_idx0-x_max)-(2*t))+1);
				u_0_0[_idx3]=0.1;
				/* _idx4 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+2)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(4*t))+1) */
				_idx4=(((_idx0+x_max)+(2*t))+1);
				u_0_0[_idx4]=0.1;
				uy_2_0[_idx2]=0.30000000000000004;
				/* _idx5 = (((((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(2*t))+1) */
				_idx5=((((_idx0+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
				u_0_0[_idx5]=0.1;
				/* _idx6 = ((((((((((((thd_idx_z+2)*x_max)+((2*t)*thd_idx_z))+(4*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(4*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(8*(t*t)))+(2*t))+1) */
				_idx6=((((_idx0+((x_max+(2*t))*y_max))+((2*t)*x_max))+(4*(t*t)))+1);
				u_0_0[_idx6]=0.1;
				uz_3_0[_idx2]=0.4;
			}
		}
	}
}

