
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(u[0][0][0][1][0]=(a*((((u[-3][0][0][0][0]+(u[0][-3][0][0][0]+u[0][0][-3][0][0]))*-2.0)+(((u[-2][0][0][0][0]+(u[0][-2][0][0][0]+u[0][0][-2][0][0]))*15.0)+((u[-1][0][0][0][0]+(u[0][-1][0][0][0]+u[0][0][-1][0][0]))*-60.0)))+((u[0][0][0][0][0]*20.0)+(((u[1][0][0][0][0]+(u[0][1][0][0][0]+u[0][0][1][0][0]))*30.0)+((u[2][0][0][0][0]+(u[0][2][0][0][0]+u[0][0][2][0][0]))*-3.0))))))

*/
__global__ void upstream_5_3d(double *  *  u_0_1_out, double *  u_0_0, double *  u_0_1, double a, int x_max, int y_max, int z_max)
{
	//double *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx10;
	int _idx11;
	int _idx12;
	int _idx13;
	int _idx14;
	int _idx15;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int _idx8;
	int _idx9;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = ((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x) */
		_idx0=((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x);
		/* _idx1 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+(((((5*p_idx_z)+15)*t)+p_idx_y)*x_max))+(((25*p_idx_z)+75)*(t*t)))+((5*p_idx_y)*t))+p_idx_x)+3) */
		_idx1=(((_idx0-(3*x_max))-(15*t))+3);
		/* _idx2 = ((((((((p_idx_z*x_max)+((5*p_idx_z)*t))*y_max)+(((((5*p_idx_z)*t)+p_idx_y)+3)*x_max))+((25*p_idx_z)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx2=((((_idx0+(((-3*x_max)-(15*t))*y_max))-((15*t)*x_max))-(75*(t*t)))+3);
		/* _idx3 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+1) */
		_idx3=(_idx0+1);
		/* _idx4 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+1)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+5)*t))+p_idx_x)+3) */
		_idx4=((_idx1+x_max)+(5*t));
		/* _idx5 = (((((((((p_idx_z+1)*x_max)+(((5*p_idx_z)+5)*t))*y_max)+((((((5*p_idx_z)+5)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+25)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx5=(((_idx2+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
		/* _idx6 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+2) */
		_idx6=(_idx3+1);
		/* _idx7 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+2)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+10)*t))+p_idx_x)+3) */
		_idx7=((_idx4+x_max)+(5*t));
		/* _idx8 = (((((((((p_idx_z+2)*x_max)+(((5*p_idx_z)+10)*t))*y_max)+((((((5*p_idx_z)+10)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+50)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx8=(((_idx5+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
		/* _idx9 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx9=(_idx3+2);
		/* _idx10 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+4) */
		_idx10=(_idx3+3);
		/* _idx11 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+4)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+20)*t))+p_idx_x)+3) */
		_idx11=((_idx9+x_max)+(5*t));
		/* _idx12 = (((((((((p_idx_z+4)*x_max)+(((5*p_idx_z)+20)*t))*y_max)+((((((5*p_idx_z)+20)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+100)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx12=(((_idx9+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
		/* _idx13 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+5) */
		_idx13=(_idx3+4);
		/* _idx14 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+5)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+25)*t))+p_idx_x)+3) */
		_idx14=((_idx11+x_max)+(5*t));
		/* _idx15 = (((((((((p_idx_z+5)*x_max)+(((5*p_idx_z)+25)*t))*y_max)+((((((5*p_idx_z)+25)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+125)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx15=(((_idx12+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
		u_0_1[_idx9]=(a*((((u_0_0[_idx0]+(u_0_0[_idx1]+u_0_0[_idx2]))*-2.0)+(((u_0_0[_idx3]+(u_0_0[_idx4]+u_0_0[_idx5]))*15.0)+((u_0_0[_idx6]+(u_0_0[_idx7]+u_0_0[_idx8]))*-60.0)))+((u_0_0[_idx9]*20.0)+(((u_0_0[_idx10]+(u_0_0[_idx11]+u_0_0[_idx12]))*30.0)+((u_0_0[_idx13]+(u_0_0[_idx14]+u_0_0[_idx15]))*-3.0)))));
	}
}

__global__ void initialize(double *  u_0_0, double *  u_0_1, double a, int x_max, int y_max, int z_max)
{
	double *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx10;
	int _idx11;
	int _idx12;
	int _idx13;
	int _idx14;
	int _idx15;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int _idx8;
	int _idx9;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = ((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x) */
		_idx0=((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x);
		u_0_0[_idx0]=0.1;
		/* _idx1 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+1) */
		_idx1=(_idx0+1);
		u_0_0[_idx1]=0.1;
		/* _idx2 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+2) */
		_idx2=(_idx1+1);
		u_0_0[_idx2]=0.1;
		/* _idx3 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+(((((5*p_idx_z)+15)*t)+p_idx_y)*x_max))+(((25*p_idx_z)+75)*(t*t)))+((5*p_idx_y)*t))+p_idx_x)+3) */
		_idx3=(((_idx1-(3*x_max))-(15*t))+2);
		u_0_0[_idx3]=0.1;
		/* _idx4 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+1)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+5)*t))+p_idx_x)+3) */
		_idx4=((_idx3+x_max)+(5*t));
		u_0_0[_idx4]=0.1;
		/* _idx5 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+2)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+10)*t))+p_idx_x)+3) */
		_idx5=((_idx4+x_max)+(5*t));
		u_0_0[_idx5]=0.1;
		/* _idx6 = ((((((((p_idx_z*x_max)+((5*p_idx_z)*t))*y_max)+(((((5*p_idx_z)*t)+p_idx_y)+3)*x_max))+((25*p_idx_z)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx6=((((_idx1+(((-3*x_max)-(15*t))*y_max))-((15*t)*x_max))-(75*(t*t)))+2);
		u_0_0[_idx6]=0.1;
		/* _idx7 = (((((((((p_idx_z+1)*x_max)+(((5*p_idx_z)+5)*t))*y_max)+((((((5*p_idx_z)+5)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+25)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx7=(((_idx6+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
		u_0_0[_idx7]=0.1;
		/* _idx8 = (((((((((p_idx_z+2)*x_max)+(((5*p_idx_z)+10)*t))*y_max)+((((((5*p_idx_z)+10)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+50)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx8=(((_idx7+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
		u_0_0[_idx8]=0.1;
		/* _idx9 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx9=(_idx1+2);
		u_0_0[_idx9]=0.1;
		/* _idx10 = (((((((((p_idx_z+4)*x_max)+(((5*p_idx_z)+20)*t))*y_max)+((((((5*p_idx_z)+20)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+100)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx10=(((_idx9+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
		u_0_0[_idx10]=0.1;
		/* _idx11 = (((((((((p_idx_z+5)*x_max)+(((5*p_idx_z)+25)*t))*y_max)+((((((5*p_idx_z)+25)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+125)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+3) */
		_idx11=(((_idx10+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
		u_0_0[_idx11]=0.1;
		/* _idx12 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+4)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+20)*t))+p_idx_x)+3) */
		_idx12=((_idx9+x_max)+(5*t));
		u_0_0[_idx12]=0.1;
		/* _idx13 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+5)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+25)*t))+p_idx_x)+3) */
		_idx13=((_idx12+x_max)+(5*t));
		u_0_0[_idx13]=0.1;
		/* _idx14 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+4) */
		_idx14=(_idx1+3);
		u_0_0[_idx14]=0.1;
		/* _idx15 = (((((((((p_idx_z+3)*x_max)+(((5*p_idx_z)+15)*t))*y_max)+((((((5*p_idx_z)+15)*t)+p_idx_y)+3)*x_max))+(((25*p_idx_z)+75)*(t*t)))+(((5*p_idx_y)+15)*t))+p_idx_x)+5) */
		_idx15=(_idx1+4);
		u_0_0[_idx15]=0.1;
		u__u_0[t][_idx9]=1.1;
	}
}

