
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1 

/*
(T[0][0][0][1][0]=((((T[0][0][0][0][0]*((c[0][0][0][0][1]*T[0][0][0][0][0])+c[0][0][0][0][2]))+c[0][0][0][0][3])+((c[0][0][0][0][4]*T[-1][0][0][0][0])+(c[0][0][0][0][5]*T[1][0][0][0][0])))+(((c[0][0][0][0][6]*T[0][-1][0][0][0])+(c[0][0][0][0][7]*T[0][1][0][0][0]))+((c[0][0][0][0][8]*T[0][0][-1][0][0])+(c[0][0][0][0][9]*T[0][0][1][0][0])))))

*/
__global__ void hyperthermia(float *  *  T_0_1_out, float *  T_0_0, float *  T_0_1, float *  c_1_0, float *  c_2_0, float *  c_3_0, float *  c_4_0, float *  c_5_0, float *  c_6_0, float *  c_7_0, float *  c_8_0, float *  c_9_0, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c)
{
/*
	const float *  const u__c_1[16] =  { c_1_0 } ;
	const float *  const u__c_2[16] =  { c_2_0 } ;
	const float *  const u__c_3[16] =  { c_3_0 } ;
	const float *  const u__c_4[16] =  { c_4_0 } ;
	const float *  const u__c_5[16] =  { c_5_0 } ;
	const float *  const u__c_6[16] =  { c_6_0 } ;
	const float *  const u__c_7[16] =  { c_7_0 } ;
	const float *  const u__c_8[16] =  { c_8_0 } ;
	const float *  const u__c_9[16] =  { c_9_0 } ;
	float *  const u__T_0[16] =  { T_0_0, T_0_1 } ;
*/
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int chunk_idx_x;
	int chunk_idx_x_max;
	int chunk_idx_y;
	int chunk_idx_y_max;
	int chunk_idx_z;
	int chunk_idx_z_max;
	int idx_1_2;
	int size_1_1;
	int size_1_2;
	//int t;
	int thd_idx_x;
	int thd_idx_y;
	int thd_idx_z;
	int thdblks_idx_x;
	int thdblks_idx_x_max;
	int thdblks_idx_y;
	int thdblks_idx_y_max;
	int thdblks_idx_z;
	int thdblks_idx_z_max;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	chunk_idx_x=(c*(threadIdx.x+(blockDim.x*blockIdx.x)));
	chunk_idx_x_max=(chunk_idx_x+c);
	chunk_idx_y=(threadIdx.y+(tmp*blockDim.y));
	chunk_idx_y_max=(chunk_idx_y+1);
	chunk_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	chunk_idx_z_max=(chunk_idx_z+1);
	thdblks_idx_x=(tbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	thdblks_idx_x_max=(thdblks_idx_x+tbx);
	thdblks_idx_y=(tby*(threadIdx.y+(tmp*blockDim.y)));
	thdblks_idx_y_max=(thdblks_idx_y+tby);
	thdblks_idx_z=(tbz*(threadIdx.z+(idx_1_2*blockDim.z)));
	thdblks_idx_z_max=(thdblks_idx_z+tbz);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in thdblks[t=t, s=(tbx, tby, tbz)][0] */
		/* Index bounds calculations for iterators in chunk[t=t, s=(c, 1, 1)][0] */
		/*
		for POINT thd[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in chunk[t=t, s=(:, :, :)][0] parallel 1 <level 2> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
			thd_idx_z=chunk_idx_z;
			thd_idx_y=chunk_idx_y;
			for (thd_idx_x=chunk_idx_x; thd_idx_x<(chunk_idx_x_max-0); thd_idx_x+=1)
			{
				/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
				/*
				u[t=(t+1), s=thd[t=?, s=?][0]][0]=stencil(u[t=t, s=thd[t=?, s=?][0]][0])
				*/
				/* _idx0 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t))+1) */
				_idx0=((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t))+1);
				/* _idx1 = ((((thd_idx_z*x_max)*y_max)+(thd_idx_y*x_max))+thd_idx_x) */
				_idx1=(((((((_idx0+(((( - x_max)-((2*t)*thd_idx_z))-(2*t))*y_max))+(((((-2*t)*thd_idx_z)-(2*t))-1)*x_max))-((4*(t*t))*thd_idx_z))-((2*t)*thd_idx_y))-(4*(t*t)))-(2*t))-1);
				/* _idx2 = (((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t)) */
				_idx2=(_idx0-1);
				/* _idx3 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t))+2) */
				_idx3=(_idx2+2);
				/* _idx4 = (((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+1) */
				_idx4=((_idx0-x_max)-(2*t));
				/* _idx5 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+2)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(4*t))+1) */
				_idx5=((_idx0+x_max)+(2*t));
				/* _idx6 = (((((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(2*t))+1) */
				_idx6=((((_idx2+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
				/* _idx7 = ((((((((((((thd_idx_z+2)*x_max)+((2*t)*thd_idx_z))+(4*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(4*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(8*(t*t)))+(2*t))+1) */
				_idx7=(((_idx0+((x_max+(2*t))*y_max))+((2*t)*x_max))+(4*(t*t)));
//				u__T_0[t][_idx0]=((((u__T_0[(t-1)][_idx0]*((u__c_1[(t-1)][_idx1]*u__T_0[(t-1)][_idx0])+u__c_2[(t-1)][_idx1]))+u__c_3[(t-1)][_idx1])+((u__c_4[(t-1)][_idx1]*u__T_0[(t-1)][_idx2])+(u__c_5[(t-1)][_idx1]*u__T_0[(t-1)][_idx3])))+(((u__c_6[(t-1)][_idx1]*u__T_0[(t-1)][_idx4])+(u__c_7[(t-1)][_idx1]*u__T_0[(t-1)][_idx5]))+((u__c_8[(t-1)][_idx1]*u__T_0[(t-1)][_idx6])+(u__c_9[(t-1)][_idx1]*u__T_0[(t-1)][_idx7]))));
				T_0_1[_idx0]=((((T_0_0[_idx0]*((c_1_0[_idx1]*T_0_0[_idx0])+c_2_0[_idx1]))+c_3_0[_idx1])+((c_4_0[_idx1]*T_0_0[_idx2])+(c_5_0[_idx1]*T_0_0[_idx3])))+(((c_6_0[_idx1]*T_0_0[_idx4])+(c_7_0[_idx1]*T_0_0[_idx5]))+((c_8_0[_idx1]*T_0_0[_idx6])+(c_9_0[_idx1]*T_0_0[_idx7]))));
			}
		}
	}
}

__global__ void initialize(float *  T_0_0, float *  T_0_1, float *  c_1_0, float *  c_2_0, float *  c_3_0, float *  c_4_0, float *  c_5_0, float *  c_6_0, float *  c_7_0, float *  c_8_0, float *  c_9_0, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c)
{
	 float *  const u__c_1[16] =  { c_1_0 } ;
	 float *  const u__c_2[16] =  { c_2_0 } ;
	 float *  const u__c_3[16] =  { c_3_0 } ;
	 float *  const u__c_4[16] =  { c_4_0 } ;
	 float *  const u__c_5[16] =  { c_5_0 } ;
	 float *  const u__c_6[16] =  { c_6_0 } ;
	 float *  const u__c_7[16] =  { c_7_0 } ;
	 float *  const u__c_8[16] =  { c_8_0 } ;
	 float *  const u__c_9[16] =  { c_9_0 } ;
	float *  const u__T_0[16] =  { T_0_0, T_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int chunk_idx_x;
	int chunk_idx_x_max;
	int chunk_idx_y;
	int chunk_idx_y_max;
	int chunk_idx_z;
	int chunk_idx_z_max;
	int idx_1_2;
	int size_1_1;
	int size_1_2;
	//int t;
	int thd_idx_x;
	int thd_idx_y;
	int thd_idx_z;
	int thdblks_idx_x;
	int thdblks_idx_x_max;
	int thdblks_idx_y;
	int thdblks_idx_y_max;
	int thdblks_idx_z;
	int thdblks_idx_z_max;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	chunk_idx_x=(c*(threadIdx.x+(blockDim.x*blockIdx.x)));
	chunk_idx_x_max=(chunk_idx_x+c);
	chunk_idx_y=(threadIdx.y+(tmp*blockDim.y));
	chunk_idx_y_max=(chunk_idx_y+1);
	chunk_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	chunk_idx_z_max=(chunk_idx_z+1);
	thdblks_idx_x=(tbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	thdblks_idx_x_max=(thdblks_idx_x+tbx);
	thdblks_idx_y=(tby*(threadIdx.y+(tmp*blockDim.y)));
	thdblks_idx_y_max=(thdblks_idx_y+tby);
	thdblks_idx_z=(tbz*(threadIdx.z+(idx_1_2*blockDim.z)));
	thdblks_idx_z_max=(thdblks_idx_z+tbz);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in thdblks[t=t, s=(tbx, tby, tbz)][0] */
		/* Index bounds calculations for iterators in chunk[t=t, s=(c, 1, 1)][0] */
		/*
		for POINT thd[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in chunk[t=t, s=(:, :, :)][0] parallel 1 <level 2> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
			thd_idx_z=chunk_idx_z;
			thd_idx_y=chunk_idx_y;
			for (thd_idx_x=chunk_idx_x; thd_idx_x<(chunk_idx_x_max-0); thd_idx_x+=1)
			{
				/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
				/*
				u[t=(t+1), s=thd[t=?, s=?][0]][0]=stencil(u[t=t, s=thd[t=?, s=?][0]][0])
				*/
				/* _idx0 = (((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t)) */
				_idx0=(((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t));
				u__T_0[(t-1)][_idx0]=0.1;
				/* _idx1 = (((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+1) */
				_idx1=(((_idx0-x_max)-(2*t))+1);
				u__T_0[(t-1)][_idx1]=0.1;
				/* _idx2 = (((((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(2*t))+1) */
				_idx2=((((_idx0+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
				u__T_0[(t-1)][_idx2]=0.1;
				/* _idx3 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t))+1) */
				_idx3=(_idx0+1);
				u__T_0[(t-1)][_idx3]=0.1;
				/* _idx4 = ((((thd_idx_z*x_max)*y_max)+(thd_idx_y*x_max))+thd_idx_x) */
				_idx4=((((((_idx2-(((2*t)*thd_idx_z)*y_max))+((((-2*t)*thd_idx_z)-1)*x_max))-((4*(t*t))*thd_idx_z))-((2*t)*thd_idx_y))-(2*t))-1);
				u__c_1[(t-1)][_idx4]=0.2;
				u__c_2[(t-1)][_idx4]=0.30000000000000004;
				u__c_3[(t-1)][_idx4]=0.4;
				u__c_4[(t-1)][_idx4]=0.5;
				u__c_5[(t-1)][_idx4]=0.6000000000000001;
				u__c_6[(t-1)][_idx4]=0.7000000000000001;
				u__c_7[(t-1)][_idx4]=0.8;
				u__c_8[(t-1)][_idx4]=0.9;
				u__c_9[(t-1)][_idx4]=1.0;
				/* _idx5 = ((((((((((((thd_idx_z+2)*x_max)+((2*t)*thd_idx_z))+(4*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(4*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(8*(t*t)))+(2*t))+1) */
				_idx5=(((_idx3+((x_max+(2*t))*y_max))+((2*t)*x_max))+(4*(t*t)));
				u__T_0[(t-1)][_idx5]=0.1;
				/* _idx6 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+2)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(4*t))+1) */
				_idx6=((_idx3+x_max)+(2*t));
				u__T_0[(t-1)][_idx6]=0.1;
				/* _idx7 = ((((((((((((thd_idx_z+1)*x_max)+((2*t)*thd_idx_z))+(2*t))*y_max)+((((((2*t)*thd_idx_z)+thd_idx_y)+(2*t))+1)*x_max))+((4*(t*t))*thd_idx_z))+((2*t)*thd_idx_y))+thd_idx_x)+(4*(t*t)))+(2*t))+2) */
				_idx7=(_idx0+2);
				u__T_0[(t-1)][_idx7]=0.1;
				u__T_0[t][_idx3]=1.1;
			}
		}
	}
}

