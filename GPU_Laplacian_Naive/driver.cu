#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "patusrt.h"

// forward_decls -->
__global__ void initialize(float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max);
__global__ void laplacian(float *  *  u_0_1_out, float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max);

// <--


int main (int argc, char** argv)
{
	int i;
	hipError_t res;
	
	// prepare grids
	// declare_grids -->
	float *  u_0_1_out;
	float *  u_0_0;
	float *  u_0_1;
	if ((argc!=4))
	{
		printf("Wrong number of parameters. Syntax:\n%s <x_max> <y_max> <z_max>\n", argv[0]);
		exit(-1);
	}
	int x_max = atoi(argv[1]);
	int y_max = atoi(argv[2]);
	int z_max = atoi(argv[3]);
	// <--
	
	// allocate_grids -->
	u_0_0=((float * )malloc(((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))));
	u_0_1=((float * )malloc(((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))));
	// <--
	
	
	// declare_GPU_grids -->
	float *  u_0_1_out_gpu;
	float *  u_0_0_gpu;
	float *  u_0_1_gpu;
	dim3 thds(1, 1, 1);
	dim3 blks(x_max, (y_max*z_max), 1);
	// <--
	
	// allocate_GPU_grids -->
	hipMalloc(((void *  * )( & u_0_1_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)));
	hipMalloc(((void *  * )( & u_0_0_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)));
	hipMalloc(((void *  * )( & u_0_1_out_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float * )));
	// <--
	
	// copy_grids_to_GPU -->
	hipMemcpy(((void * )u_0_1_gpu), ((void * )u_0_1), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )u_0_0_gpu), ((void * )u_0_0), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)), hipMemcpyHostToDevice);
	// <--
	
	
	// initialize_grids -->
	initialize<<<blks, thds>>>(u_0_0_gpu, u_0_1_gpu, x_max, y_max, z_max);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Initialization]: %s.\n", hipGetErrorString (res));
	}
	
	long nFlopsPerStencil = 7;
	long nGridPointsCount = 5 * ((x_max*y_max)*z_max);
	long nBytesTransferred = 5 * (((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))+(((x_max*y_max)*z_max)*sizeof (float)));
	
	// warm up
	// compute_stencil -->
	laplacian<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, x_max, y_max, z_max);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Stencil]: %s.\n", hipGetErrorString (res));
	}
	
	// run the benchmark
	tic ();
	for (i = 0; i < 5; i++)
	{
		// compute_stencil -->
		laplacian<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, x_max, y_max, z_max);
		// <--
		
		hipDeviceSynchronize ();
	}
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);
	
	// free memory
	// deallocate_grids -->
	hipFree(((void * )u_0_1_gpu));
	hipFree(((void * )u_0_0_gpu));
	hipFree(((void * )u_0_1_out_gpu));
	free(u_0_0);
	free(u_0_1);
	// <--
	
	
	hipDeviceReset ();
	return EXIT_SUCCESS;
}
