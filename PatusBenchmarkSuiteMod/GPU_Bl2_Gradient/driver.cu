#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "patusrt.h"

// forward_decls -->
__global__ void initialize(float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c);
__global__ void gradient(float *  *  ux_1_0_out, float *  *  uy_2_0_out, float *  *  uz_3_0_out, float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c);

// <--


int main (int argc, char** argv)
{
	int i;
	hipError_t res;
	
	// prepare grids
	// declare_grids -->
	float *  ux_1_0_out;
	float *  uy_2_0_out;
	float *  uz_3_0_out;
	float *  u_0_0;
	float *  ux_1_0;
	float *  uy_2_0;
	float *  uz_3_0;
	if ((argc!=8))
	{
		printf("Wrong number of parameters. Syntax:\n%s <x_max> <y_max> <z_max> <tbx> <tby> <tbz> <c>\n", argv[0]);
		exit(-1);
	}
	int x_max = atoi(argv[1]);
	int y_max = atoi(argv[2]);
	int z_max = atoi(argv[3]);
	int tbx = atoi(argv[4]);
	int tby = atoi(argv[5]);
	int tbz = atoi(argv[6]);
	int c = atoi(argv[7]);
	// <--
	
	// allocate_grids -->
	u_0_0=((float * )malloc(((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))));
	ux_1_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	uy_2_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	uz_3_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	// <--
	
	
	// declare_GPU_grids -->
	float *  ux_1_0_out_gpu;
	float *  uy_2_0_out_gpu;
	float *  uz_3_0_out_gpu;
	float *  u_0_0_gpu;
	float *  ux_1_0_gpu;
	float *  uy_2_0_gpu;
	float *  uz_3_0_gpu;
	dim3 thds(tbx, tby, tbz);
	dim3 blks((x_max/tbx), ((y_max*z_max)/(tby*tbz)), 1);
	// <--
	
	// allocate_GPU_grids -->
	hipMalloc(((void *  * )( & uz_3_0_out_gpu)), (((x_max*y_max)*z_max)*sizeof (float * )));
	hipMalloc(((void *  * )( & uy_2_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & u_0_0_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)));
	hipMalloc(((void *  * )( & ux_1_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & uy_2_0_out_gpu)), (((x_max*y_max)*z_max)*sizeof (float * )));
	hipMalloc(((void *  * )( & uz_3_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & ux_1_0_out_gpu)), (((x_max*y_max)*z_max)*sizeof (float * )));
	// <--
	
	// copy_grids_to_GPU -->
	hipMemcpy(((void * )uy_2_0_gpu), ((void * )uy_2_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )u_0_0_gpu), ((void * )u_0_0), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )ux_1_0_gpu), ((void * )ux_1_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )uz_3_0_gpu), ((void * )uz_3_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	// <--
	
	
	// initialize_grids -->
	initialize<<<blks, thds>>>(u_0_0_gpu, ux_1_0_gpu, uy_2_0_gpu, uz_3_0_gpu, 0.1, 0.2, 0.30000000000000004, x_max, y_max, z_max, tbx, tby, tbz, c);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Initialization]: %s.\n", hipGetErrorString (res));
	}
	
	long nFlopsPerStencil = 6;
	long nGridPointsCount = 5 * ((x_max*y_max)*z_max);
	long nBytesTransferred = 5 * (((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))+(((((x_max*y_max)*z_max)*sizeof (float))+(((x_max*y_max)*z_max)*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float))));
	
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(gradient), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(gradient), hipFuncCachePreferL1);
	
	// warm up
	// compute_stencil -->
	gradient<<<blks, thds>>>(( & ux_1_0_out_gpu), ( & uy_2_0_out_gpu), ( & uz_3_0_out_gpu), u_0_0_gpu, ux_1_0_gpu, uy_2_0_gpu, uz_3_0_gpu, 0.4, 0.5, 0.6, x_max, y_max, z_max, tbx, tby, tbz, c);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Stencil]: %s.\n", hipGetErrorString (res));
	}
	
	// run the benchmark
	tic ();
	for (i = 0; i < 5; i++)
	{
		// compute_stencil -->
		gradient<<<blks, thds>>>(( & ux_1_0_out_gpu), ( & uy_2_0_out_gpu), ( & uz_3_0_out_gpu), u_0_0_gpu, ux_1_0_gpu, uy_2_0_gpu, uz_3_0_gpu, 0.7, 0.7999999999999999, 0.8999999999999999, x_max, y_max, z_max, tbx, tby, tbz, c);
		// <--
		
		hipDeviceSynchronize ();
	}
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);
	
	// free memory
	// deallocate_grids -->
	hipFree(((void * )uz_3_0_out_gpu));
	hipFree(((void * )uy_2_0_gpu));
	hipFree(((void * )u_0_0_gpu));
	hipFree(((void * )ux_1_0_gpu));
	hipFree(((void * )uy_2_0_out_gpu));
	hipFree(((void * )uz_3_0_gpu));
	hipFree(((void * )ux_1_0_out_gpu));
	free(u_0_0);
	free(ux_1_0);
	free(uy_2_0);
	free(uz_3_0);
	// <--
	
	
	hipDeviceReset ();
	return EXIT_SUCCESS;
}
