#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "patusrt.h"

// forward_decls -->
__global__ void initialize(double *  u_0_0, double *  u_0_1, double a, int x_max, int y_max, int z_max);
__global__ void upstream_5_3d(double *  *  u_0_1_out, double *  u_0_0, double *  u_0_1, double a, int x_max, int y_max, int z_max);

// <--


int main (int argc, char** argv)
{
	int i;
	hipError_t res;
	
	// prepare grids
	// declare_grids -->
	double *  u_0_1_out;
	double *  u_0_0;
	double *  u_0_1;
	if ((argc!=4))
	{
		printf("Wrong number of parameters. Syntax:\n%s <x_max> <y_max> <z_max>\n", argv[0]);
		exit(-1);
	}
	int x_max = atoi(argv[1]);
	int y_max = atoi(argv[2]);
	int z_max = atoi(argv[3]);
	// <--
	
	// allocate_grids -->
	u_0_0=((double * )malloc(((((x_max+10)*(y_max+10))*(z_max+10))*sizeof (double))));
	u_0_1=((double * )malloc(((((x_max+10)*(y_max+10))*(z_max+10))*sizeof (double))));
	// <--
	
	
	// declare_GPU_grids -->
	double *  u_0_1_out_gpu;
	double *  u_0_0_gpu;
	double *  u_0_1_gpu;
	dim3 thds(1, 1, 1);
	dim3 blks(x_max, (y_max*z_max), 1);
	// <--
	
	// allocate_GPU_grids -->
	hipMalloc(((void *  * )( & u_0_1_gpu)), ((((x_max+10)*(y_max+10))*(z_max+10))*sizeof (double)));
	hipMalloc(((void *  * )( & u_0_0_gpu)), ((((x_max+10)*(y_max+10))*(z_max+10))*sizeof (double)));
	hipMalloc(((void *  * )( & u_0_1_out_gpu)), ((((x_max+10)*(y_max+10))*(z_max+10))*sizeof (double * )));
	// <--
	
	// copy_grids_to_GPU -->
	hipMemcpy(((void * )u_0_1_gpu), ((void * )u_0_1), ((((x_max+10)*(y_max+10))*(z_max+10))*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )u_0_0_gpu), ((void * )u_0_0), ((((x_max+10)*(y_max+10))*(z_max+10))*sizeof (double)), hipMemcpyHostToDevice);
	// <--
	
	
	// initialize_grids -->
	initialize<<<blks, thds>>>(u_0_0_gpu, u_0_1_gpu, 0.1, x_max, y_max, z_max);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Initialization]: %s.\n", hipGetErrorString (res));
	}
	
	long nFlopsPerStencil = 22;
	long nGridPointsCount = 5 * ((x_max*y_max)*z_max);
	long nBytesTransferred = 5 * (((((x_max+10)*(y_max+10))*(z_max+10))*sizeof (double))+(((x_max*y_max)*z_max)*sizeof (double)));
	
	// warm up
	// compute_stencil -->
	upstream_5_3d<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, 0.2, x_max, y_max, z_max);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Stencil]: %s.\n", hipGetErrorString (res));
	}
	
	// run the benchmark
	tic ();
	for (i = 0; i < 5; i++)
	{
		// compute_stencil -->
		upstream_5_3d<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, 0.30000000000000004, x_max, y_max, z_max);
		// <--
		
		hipDeviceSynchronize ();
	}
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);
	
	// free memory
	// deallocate_grids -->
	hipFree(((void * )u_0_1_gpu));
	hipFree(((void * )u_0_0_gpu));
	hipFree(((void * )u_0_1_out_gpu));
	free(u_0_0);
	free(u_0_1);
	// <--
	
	
	hipDeviceReset ();
	return EXIT_SUCCESS;
}
