#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "patusrt.h"

// forward_decls -->
__global__ void initialize(float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c);
__global__ void laplacian(float *  *  u_0_1_out, float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c);
void initialize_cpu(float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max);
void laplacian_cpu(float *  *  u_0_1_out, float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max, int t_max);
// <--


int main (int argc, char** argv)
{
	int i;
	hipError_t res;
	
	// prepare grids
	// declare_grids -->
	float *  u_0_1_out;
	float *  u_0_0;
	float *  u_0_1;
	float * u_0_1_out_host;
	if ((argc!=8))
	{
		printf("Wrong number of parameters. Syntax:\n%s <x_max> <y_max> <z_max> <tbx> <tby> <tbz> <c>\n", argv[0]);
		exit(-1);
	}
	int x_max = atoi(argv[1]);
	int y_max = atoi(argv[2]);
	int z_max = atoi(argv[3]);
	int tbx = atoi(argv[4]);
	int tby = atoi(argv[5]);
	int tbz = atoi(argv[6]);
	int c = atoi(argv[7]);
	// <--
	
	// allocate_grids -->
	u_0_0=((float * )malloc(((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))));
	u_0_1=((float * )malloc(((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))));
	u_0_1_out_host = ((float * )malloc(((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))));
	
	// <--
	
	
	// declare_GPU_grids -->
	float *  u_0_1_out_gpu;
	float *  u_0_0_gpu;
	float *  u_0_1_gpu;

	dim3 thds(tbx, tby, tbz);
	dim3 blks((x_max/tbx), ((y_max*z_max)/(tby*tbz)), 1);
	// <--
	
	// allocate_GPU_grids -->
	hipMalloc(((void *  * )( & u_0_1_out_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float * )));
	hipMalloc(((void *  * )( & u_0_0_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)));
	hipMalloc(((void *  * )( & u_0_1_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)));
	// <--
	
	
	
	// initialize_grids on host-->
	initialize_cpu(u_0_0, u_0_1, x_max, y_max, z_max);
	/* initialize<<<blks, thds>>>(u_0_0_gpu, u_0_1_gpu, x_max, y_max, z_max, tbx, tby, tbz, c); */
	// <--

	
	// copy_grids_to_GPU -->
	hipMemcpy(((void * )u_0_0_gpu), ((void * )u_0_0), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )u_0_1_gpu), ((void * )u_0_1), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)), hipMemcpyHostToDevice);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Initialization]: %s.\n", hipGetErrorString (res));
	}
	
	long nFlopsPerStencil = 7;
	int iterations = 100;
	long nGridPointsCount = iterations * ((x_max*y_max)*z_max);
	long nBytesTransferred = iterations * (((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))+(((x_max*y_max)*z_max)*sizeof (float)));
	
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(laplacian), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(laplacian), hipFuncCachePreferL1);
	
	// warm up
	// compute_stencil -->
	laplacian<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, x_max, y_max, z_max, tbx, tby, tbz, c);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Stencil]: %s.\n", hipGetErrorString (res));
	}
	
	// run the benchmark
	tic ();
	for (i = 0; i < iterations; i++)
	{
		// compute_stencil -->
		laplacian<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, x_max, y_max, z_max, tbx, tby, tbz, c);
		// <--
		
		hipDeviceSynchronize ();
	}
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);

	hipMemcpy(((void * )u_0_1_out_host), ((void * )u_0_1_out_gpu), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)), hipMemcpyDeviceToHost);


	//warm up
	laplacian_cpu(( & u_0_1_out), u_0_0, u_0_1, x_max, y_max, z_max,iterations);
	tic ();
	// compute_stencil -->
	laplacian_cpu(( & u_0_1_out), u_0_0, u_0_1, x_max, y_max, z_max,iterations);
	// <--
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);

	int error_count=0;
	int x,y,z;
	for(y=1;y<x_max+1;y++) {
	  for(x=1;x<x_max+1;x++) {
	    for(z=1;z<y_max+1;z++) {
	      i = x + (x_max+2)*y + (x_max+2)*(y_max+2)*z;
	      if(fabs(u_0_1_out_host[i] - u_0_1[i])>0.001) {
		error_count++;
		printf("%dth error encountered at u[%d]: |%f-%f|=%5.16f\n",error_count,i,u_0_1_out_host[i],u_0_1[i],fabs(u_0_1_out_host[i] - u_0_1[i]));
		if(error_count>30) {
		  printf("too many errors\n"); exit(1);
		}
	      }
	    }
	  }
	}
	
	
	printf("Error Check Successful. No errors encountered.\n");	  
	/* saving results */
	FILE* fp;
	char filename[100];
	sprintf(filename,"/users/rietmann/tmp/stencil_laplacian_cuda_%.3d_%.3d_%.3d.dat",x_max,y_max,z_max);
	fp = fopen(filename,"w");
	if(!fp) {printf("couldn't open file: %s\n",filename); exit(1);}
	fprintf(fp,"%d\n",x_max);
	fprintf(fp,"%d\n",y_max);
	fprintf(fp,"%d\n",z_max);	  
	for(i=0;i<(x_max+4)*(y_max+4)*(z_max+4);i++) {
	  fprintf(fp,"%2.6f\n", u_0_1_out_host[i]);
	}
	fclose(fp);
	
	// free memory
	// deallocate_grids -->
	hipFree(((void * )u_0_1_out_gpu));
	hipFree(((void * )u_0_0_gpu));
	hipFree(((void * )u_0_1_gpu));
	free(u_0_0);
	free(u_0_1);
	// <--
	
	
	hipDeviceReset ();
	return EXIT_SUCCESS;
}

void initialize_cpu(float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max)
{
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int p_idx_x;
	int p_idx_y;
	int p_idx_z;
	int t;
	float *  __restrict__ const u__u_0[2] =  { u_0_0, u_0_1 } ;
	/*
	Initializations
	*/
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/

	{
		/*
		for POINT p[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in u[t=t, s=(:, :, :)][0] parallel 1 <level 0> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
			for (p_idx_z=0; p_idx_z<z_max; p_idx_z+=1)
			{
				for (p_idx_y=0; p_idx_y<y_max; p_idx_y+=1)
				{
					for (p_idx_x=0; p_idx_x<x_max; p_idx_x+=1)
					{
						/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
						/*
						u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
						*/
						/* _idx0 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+3)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+6) */
						_idx0=((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+3)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+6);
						u__u_0[0][_idx0]=0.1;
						/* _idx1 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+2)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+5) */
						_idx1=((_idx0-x_max)-1);
						u__u_0[0][_idx1]=0.1;
						/* _idx2 = ((((((((p_idx_z*x_max)+(2*p_idx_z))*y_max)+((((2*p_idx_z)+p_idx_y)+1)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+3) */
						_idx2=(((_idx1+((( - x_max)-2)*y_max))-x_max)-2);
						u__u_0[0][_idx2]=0.1;
						/* _idx3 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+3)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+7) */
						_idx3=((_idx1+x_max)+2);
						u__u_0[0][_idx3]=0.1;
						/* _idx4 = ((((((((((p_idx_z+2)*x_max)+(2*p_idx_z))+4)*y_max)+((((2*p_idx_z)+p_idx_y)+5)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+11) */
						_idx4=(((_idx1+((x_max+2)*y_max))+(3*x_max))+6);
						u__u_0[0][_idx4]=0.1;
						/* _idx5 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+4)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+9) */
						_idx5=((_idx0+x_max)+3);
						u__u_0[0][_idx5]=0.1;
						/* _idx6 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+3)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+8) */
						_idx6=((_idx1+x_max)+3);
						u__u_0[0][_idx6]=0.1;
						u__u_0[1][_idx3]=1.1;
					}
				}
			}
		}
	}
}

void laplacian_cpu(float *  *  u_0_1_out, float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max, int t_max)
{
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int p_idx_x;
	int p_idx_y;
	int p_idx_z;
	int t;

	/*
	Initializations
	*/
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/

	{
	  int count=0;
	  for (t=1; t<=t_max; t+=1)
	    {
			/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
			for (p_idx_z=0; p_idx_z<z_max; p_idx_z+=1)
			{
				for (p_idx_y=0; p_idx_y<y_max; p_idx_y+=1)
				{
					for (p_idx_x=0; p_idx_x<x_max; p_idx_x+=1)
					{
						/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
						/*
						u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
						*/
						/* _idx0 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+3)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+8) */
						_idx0=((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+3)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+8);
						/* _idx1 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+3)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+6) */
						_idx1=(_idx0-2);
						/* _idx2 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+4)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+9) */
						_idx2=((_idx1+x_max)+3);
						/* _idx3 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+2)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+5) */
						_idx3=((_idx1-x_max)-1);
						/* _idx4 = ((((((((((p_idx_z+2)*x_max)+(2*p_idx_z))+4)*y_max)+((((2*p_idx_z)+p_idx_y)+5)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+11) */
						_idx4=(((_idx3+((x_max+2)*y_max))+(3*x_max))+6);
						/* _idx5 = ((((((((p_idx_z*x_max)+(2*p_idx_z))*y_max)+((((2*p_idx_z)+p_idx_y)+1)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+3) */
						_idx5=(((_idx3+((( - x_max)-2)*y_max))-x_max)-2);
						/* _idx6 = ((((((((((p_idx_z+1)*x_max)+(2*p_idx_z))+2)*y_max)+((((2*p_idx_z)+p_idx_y)+3)*x_max))+(4*p_idx_z))+(2*p_idx_y))+p_idx_x)+7) */
						_idx6=((_idx3+x_max)+2);
						u_0_1[_idx6]=((((u_0_0[_idx0]+(u_0_0[_idx1]+u_0_0[_idx2]))+(u_0_0[_idx3]+(u_0_0[_idx4]+u_0_0[_idx5])))*0.25)-u_0_0[_idx6]);
						
						
						/* debugging */
						/* u_0_0[count] = p_idx_x + p_idx_y*x_max + p_idx_z*x_max*y_max; */
						/* count++; */
						
						/* u_0_1[p_idx_x + p_idx_y*x_max + p_idx_z*x_max*y_max] = _idx3; */
						/* u_0_1[p_idx_x + p_idx_y*x_max + p_idx_z*x_max] = p_idx_x + p_idx_y*x_max + p_idx_z*x_max; */
						/* end debugging */
						
						/* printf("u_0_0[_idx0]=%f,u_0_0[_idx2]=%f,u_0_0[_idx1]=%f\n",u_0_0[_idx0],u_0_0[_idx2],u_0_0[_idx1]); */
						
					}
				}
			}
		}
	}
	*u_0_1_out = u_0_1;
}
