#include "hip/hip_runtime.h"
/* CUDA_SAFE_CALL (hipMalloc ((void**) &u0_device, Nx*sizeof (hipComplex))); */
/* nls_stencil<<< blocks, threads >>> (u_device,u0_device); */
/* CUT_DEVICE_INIT (argc, argv); */
/* CUDA_SAFE_CALL (hipMemcpy (u,u_device, N*NUMBER_OF_DOMAINS*sizeof (hipComplex), */
/* 				  hipMemcpyDeviceToHost)); */

__global__ void laplacian_gpu(float *  *  u_0_1_out, float * u_0_0, float * u_0_1, int x_max, int y_max, int z_max)
{
  int _idx0;
  int _idx1;
  int _idx2;
  int _idx3;
  int _idx4;
  int _idx5;
  int _idx6;
  int p_idx_x;
  int p_idx_y;
  int p_idx_z;
  //	int t;
  //float *  __restrict__ const u__u_0[16] =  { u_0_0, u_0_1 } ;
  int u_idx_x;
  int u_idx_x_max;
  int u_idx_y;
  int u_idx_y_max;
  int u_idx_z;
  int u_idx_z_max;

  __shared__ float us[BLOCKx*BLOCKy*BLOCKz];

  id = 
  
  us[

  
      /* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
      /* for (p_idx_z=0; p_idx_z<z_max; p_idx_z+=1) */
      /* 	{ */
      /* 	  for (p_idx_y=0; p_idx_y<y_max; p_idx_y+=1) */
      /* 	    { */
      /* 	      for (p_idx_x=0; p_idx_x<x_max; p_idx_x+=1) */
      /* 		{ */
		  
  _idx0=(((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)))+(((2*p_idx_y)+2)))+p_idx_x)+2);
  _idx1=_idx0-2;
  _idx2=(((_idx1+x_max)+2)+1);
  _idx3=(((_idx1-x_max)-2)+1);
  _idx4=((((_idx3+((x_max+2)*y_max))+((2+1)*x_max))+4)+2);
  _idx5=((((_idx1+((( -x_max)-2)*y_max))-(2*x_max))-4)+1);
  _idx6=(_idx1+1);

  u_0_1[_idx6]=((((u_0_0[_idx0]+(u_0_0[_idx1]+u_0_0[_idx2]))+(u_0_0[_idx3]+(u_0_0[_idx4]+u_0_0[_idx5])))*0.25)-u_0_0[_idx6]);
}
}
}
    
  
  *u_0_1_out = u_0_1;
}

