
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(u[0][0][0][0][0]=((alpha*(ux[1][0][0][0][1]-ux[-1][0][0][0][1]))+((beta*(uy[0][1][0][0][2]-uy[0][-1][0][0][2]))+(gamma*(uz[0][0][1][0][3]-uz[0][0][-1][0][3])))))

*/
__global__ void divergence(float *  *  u_0_0_out, float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max)
{
/*
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
*/

	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	//for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = (((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(p_idx_y*x_max))+((2*p_idx_y)*t))+p_idx_x)+2) */
		_idx0=(((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(p_idx_y*x_max))+((2*p_idx_y)*t))+p_idx_x)+2);
		/* _idx1 = ((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(p_idx_y*x_max))+((2*p_idx_y)*t))+p_idx_x) */
		_idx1=(_idx0-2);
		/* _idx2 = ((((p_idx_z*x_max)*y_max)+(((((2*p_idx_z)*t)+p_idx_y)+2)*x_max))+p_idx_x) */
		_idx2=(((_idx1-(((2*p_idx_z)*t)*y_max))+((((2*p_idx_z)*t)+2)*x_max))-((2*p_idx_y)*t));
		/* _idx3 = ((((p_idx_z*x_max)*y_max)+((((2*p_idx_z)*t)+p_idx_y)*x_max))+p_idx_x) */
		_idx3=(_idx2-(2*x_max));
		/* _idx4 = (((((p_idx_z+2)*x_max)*y_max)+(p_idx_y*x_max))+p_idx_x) */
		_idx4=((_idx3+((2*x_max)*y_max))-(((2*p_idx_z)*t)*x_max));
		/* _idx5 = ((((p_idx_z*x_max)*y_max)+(p_idx_y*x_max))+p_idx_x) */
		_idx5=(_idx4-((2*x_max)*y_max));
		u_0_0[_idx5]=((alpha*(ux_1_0[_idx0]-ux_1_0[_idx1]))+((beta*(uy_2_0[_idx2]-uy_2_0[_idx3]))+(gamma*(uz_3_0[_idx4]-uz_3_0[_idx5]))));
	}
}

__global__ void initialize(float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max)
{
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = ((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(p_idx_y*x_max))+((2*p_idx_y)*t))+p_idx_x) */
		_idx0=((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(p_idx_y*x_max))+((2*p_idx_y)*t))+p_idx_x);
		ux_1_0[_idx0]=0.2;
		/* _idx1 = ((((p_idx_z*x_max)*y_max)+((((2*p_idx_z)*t)+p_idx_y)*x_max))+p_idx_x) */
		_idx1=(((_idx0-(((2*p_idx_z)*t)*y_max))+(((2*p_idx_z)*t)*x_max))-((2*p_idx_y)*t));
		uy_2_0[_idx1]=0.30000000000000004;
		/* _idx2 = ((((p_idx_z*x_max)*y_max)+(p_idx_y*x_max))+p_idx_x) */
		_idx2=(_idx1-(((2*p_idx_z)*t)*x_max));
		uz_3_0[_idx2]=0.4;
		/* _idx3 = (((((p_idx_z+2)*x_max)*y_max)+(p_idx_y*x_max))+p_idx_x) */
		_idx3=(_idx2+((2*x_max)*y_max));
		uz_3_0[_idx3]=0.4;
		/* _idx4 = ((((p_idx_z*x_max)*y_max)+(((((2*p_idx_z)*t)+p_idx_y)+2)*x_max))+p_idx_x) */
		_idx4=(_idx1+(2*x_max));
		uy_2_0[_idx4]=0.30000000000000004;
		/* _idx5 = (((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(p_idx_y*x_max))+((2*p_idx_y)*t))+p_idx_x)+2) */
		_idx5=(_idx0+2);
		ux_1_0[_idx5]=0.2;
		u_0_0[_idx2]=0.1;
	}
}

