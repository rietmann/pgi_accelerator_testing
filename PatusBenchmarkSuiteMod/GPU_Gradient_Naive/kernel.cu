
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(ux[0][0][0][0][1]=(alpha*(u[1][0][0][0][0]+u[-1][0][0][0][0])))
(uy[0][0][0][0][2]=(beta*(u[0][1][0][0][0]+u[0][-1][0][0][0])))
(uz[0][0][0][0][3]=(gamma*(u[0][0][1][0][0]+u[0][0][-1][0][0])))

*/
__global__ void gradient(float *  *  ux_1_0_out, float *  *  uy_2_0_out, float *  *  uz_3_0_out, float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max)
{
/*
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
	*/
	
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+2) */
		_idx0=(((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+2);
		/* _idx1 = ((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x) */
		_idx1=(_idx0-2);
		/* _idx2 = ((((p_idx_z*x_max)*y_max)+(p_idx_y*x_max))+p_idx_x) */
		_idx2=((((_idx1+(((((-2*p_idx_z)-2)*t)-x_max)*y_max))+(((((-2*p_idx_z)-2)*t)-1)*x_max))+(((-4*p_idx_z)-4)*(t*t)))+(((-2*p_idx_y)-2)*t));
		ux_1_0[_idx2]=(alpha*(u_0_0[_idx0]+u_0_0[_idx1]));
		/* _idx3 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+2)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+4)*t))+p_idx_x)+1) */
		_idx3=(((_idx1+x_max)+(2*t))+1);
		/* _idx4 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+(((((2*p_idx_z)+2)*t)+p_idx_y)*x_max))+(((4*p_idx_z)+4)*(t*t)))+((2*p_idx_y)*t))+p_idx_x)+1) */
		_idx4=(((_idx1-x_max)-(2*t))+1);
		uy_2_0[_idx2]=(beta*(u_0_0[_idx3]+u_0_0[_idx4]));
		/* _idx5 = (((((((((p_idx_z+2)*x_max)+(((2*p_idx_z)+4)*t))*y_max)+((((((2*p_idx_z)+4)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+8)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx5=((((_idx4+((x_max+(2*t))*y_max))+(((2*t)+1)*x_max))+(4*(t*t)))+(2*t));
		/* _idx6 = ((((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(((((2*p_idx_z)*t)+p_idx_y)+1)*x_max))+((4*p_idx_z)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx6=((((_idx1+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
		uz_3_0[_idx2]=(gamma*(u_0_0[_idx5]+u_0_0[_idx6]));
	}
}

__global__ void initialize(float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max)
{
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = ((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x) */
		_idx0=((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x);
		u_0_0[_idx0]=0.1;
		/* _idx1 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+2) */
		_idx1=(_idx0+2);
		u_0_0[_idx1]=0.1;
		/* _idx2 = ((((p_idx_z*x_max)*y_max)+(p_idx_y*x_max))+p_idx_x) */
		_idx2=((((_idx0+(((((-2*p_idx_z)-2)*t)-x_max)*y_max))+(((((-2*p_idx_z)-2)*t)-1)*x_max))+(((-4*p_idx_z)-4)*(t*t)))+(((-2*p_idx_y)-2)*t));
		u__ux_1[(t-1)][_idx2]=0.2;
		/* _idx3 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+(((((2*p_idx_z)+2)*t)+p_idx_y)*x_max))+(((4*p_idx_z)+4)*(t*t)))+((2*p_idx_y)*t))+p_idx_x)+1) */
		_idx3=(((_idx0-x_max)-(2*t))+1);
		u_0_0[_idx3]=0.1;
		/* _idx4 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+2)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+4)*t))+p_idx_x)+1) */
		_idx4=((_idx3+(2*x_max))+(4*t));
		u_0_0[_idx4]=0.1;
		u__uy_2[(t-1)][_idx2]=0.30000000000000004;
		/* _idx5 = ((((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(((((2*p_idx_z)*t)+p_idx_y)+1)*x_max))+((4*p_idx_z)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx5=((((_idx0+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
		u_0_0[_idx5]=0.1;
		/* _idx6 = (((((((((p_idx_z+2)*x_max)+(((2*p_idx_z)+4)*t))*y_max)+((((((2*p_idx_z)+4)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+8)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx6=((((_idx3+((x_max+(2*t))*y_max))+(((2*t)+1)*x_max))+(4*(t*t)))+(2*t));
		u_0_0[_idx6]=0.1;
		u__uz_3[(t-1)][_idx2]=0.4;
	}
}

