#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "patusrt.h"

// forward_decls -->
__global__ void initialize(double *  u_0_0, double *  u_0_1, double *  a_1_0, double *  b_2_0, double *  c_3_0, int x_max, int y_max, int z_max, int cbx);
__global__ void tricubic_interpolation(double *  *  u_0_1_out, double *  u_0_0, double *  u_0_1, double *  a_1_0, double *  b_2_0, double *  c_3_0, int x_max, int y_max, int z_max, int cbx);

// <--


int main (int argc, char** argv)
{
	int i;
	hipError_t res;
	
	// prepare grids
	// declare_grids -->
	double *  u_0_1_out;
	double *  u_0_0;
	double *  u_0_1;
	double *  a_1_0;
	double *  b_2_0;
	double *  c_3_0;
	if ((argc!=5))
	{
		printf("Wrong number of parameters. Syntax:\n%s <x_max> <y_max> <z_max> <cbx>\n", argv[0]);
		exit(-1);
	}
	int x_max = atoi(argv[1]);
	int y_max = atoi(argv[2]);
	int z_max = atoi(argv[3]);
	int cbx = atoi(argv[4]);
	// <--
	
	// allocate_grids -->
	u_0_0=((double * )malloc(((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double))));
	u_0_1=((double * )malloc(((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double))));
	a_1_0=((double * )malloc((((x_max*y_max)*z_max)*sizeof (double))));
	b_2_0=((double * )malloc((((x_max*y_max)*z_max)*sizeof (double))));
	c_3_0=((double * )malloc((((x_max*y_max)*z_max)*sizeof (double))));
	// <--
	
	
	// declare_GPU_grids -->
	double *  u_0_1_out_gpu;
	double *  u_0_0_gpu;
	double *  u_0_1_gpu;
	double *  a_1_0_gpu;
	double *  b_2_0_gpu;
	double *  c_3_0_gpu;
	dim3 thds(cbx, 1, 1);
	dim3 blks((x_max/cbx), (y_max*z_max), 1);
	// <--
	
	// allocate_GPU_grids -->
	hipMalloc(((void *  * )( & a_1_0_gpu)), (((x_max*y_max)*z_max)*sizeof (double)));
	hipMalloc(((void *  * )( & c_3_0_gpu)), (((x_max*y_max)*z_max)*sizeof (double)));
	hipMalloc(((void *  * )( & u_0_0_gpu)), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)));
	hipMalloc(((void *  * )( & b_2_0_gpu)), (((x_max*y_max)*z_max)*sizeof (double)));
	hipMalloc(((void *  * )( & u_0_1_out_gpu)), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double * )));
	hipMalloc(((void *  * )( & u_0_1_gpu)), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)));
	// <--
	
	// copy_grids_to_GPU -->
	hipMemcpy(((void * )a_1_0_gpu), ((void * )a_1_0), (((x_max*y_max)*z_max)*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_3_0_gpu), ((void * )c_3_0), (((x_max*y_max)*z_max)*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )u_0_0_gpu), ((void * )u_0_0), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )b_2_0_gpu), ((void * )b_2_0), (((x_max*y_max)*z_max)*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )u_0_1_gpu), ((void * )u_0_1), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)), hipMemcpyHostToDevice);
	// <--
	
	
	// initialize_grids -->
	initialize<<<blks, thds>>>(u_0_0_gpu, u_0_1_gpu, a_1_0_gpu, b_2_0_gpu, c_3_0_gpu, x_max, y_max, z_max, cbx);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Initialization]: %s.\n", hipGetErrorString (res));
	}
	
	long nFlopsPerStencil = 318;
	long nGridPointsCount = 5 * ((x_max*y_max)*z_max);
	long nBytesTransferred = 5 * (((((((x_max*y_max)*z_max)*sizeof (double))+(((x_max*y_max)*z_max)*sizeof (double)))+((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)))+(((x_max*y_max)*z_max)*sizeof (double)))+(((x_max*y_max)*z_max)*sizeof (double)));
	
	// warm up
	// compute_stencil -->
	tricubic_interpolation<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, a_1_0_gpu, b_2_0_gpu, c_3_0_gpu, x_max, y_max, z_max, cbx);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Stencil]: %s.\n", hipGetErrorString (res));
	}
	
	// run the benchmark
	tic ();
	for (i = 0; i < 5; i++)
	{
		// compute_stencil -->
		tricubic_interpolation<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, a_1_0_gpu, b_2_0_gpu, c_3_0_gpu, x_max, y_max, z_max, cbx);
		// <--
		
		hipDeviceSynchronize ();
	}
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);
	
	// free memory
	// deallocate_grids -->
	hipFree(((void * )a_1_0_gpu));
	hipFree(((void * )c_3_0_gpu));
	hipFree(((void * )u_0_0_gpu));
	hipFree(((void * )b_2_0_gpu));
	hipFree(((void * )u_0_1_out_gpu));
	hipFree(((void * )u_0_1_gpu));
	free(u_0_0);
	free(u_0_1);
	free(a_1_0);
	free(b_2_0);
	free(c_3_0);
	// <--
	
	
	hipDeviceReset ();
	return EXIT_SUCCESS;
}
