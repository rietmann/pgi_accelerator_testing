#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "patusrt.h"

// forward_decls -->
__global__ void initialize(double *  u_0_0, double *  u_0_1, double *  a_1_0, double *  b_2_0, double *  c_3_0, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c);
__global__ void tricubic_interpolation(double *  *  u_0_1_out, double *  u_0_0, double *  u_0_1, double *  a_1_0, double *  b_2_0, double *  c_3_0, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c);

// <--


int main (int argc, char** argv)
{
	int i;
	hipError_t res;
	
	// prepare grids
	// declare_grids -->
	double *  u_0_1_out;
	double *  u_0_0;
	double *  u_0_1;
	double *  a_1_0;
	double *  b_2_0;
	double *  c_3_0;
	if ((argc!=8))
	{
		printf("Wrong number of parameters. Syntax:\n%s <x_max> <y_max> <z_max> <tbx> <tby> <tbz> <c>\n", argv[0]);
		exit(-1);
	}
	int x_max = atoi(argv[1]);
	int y_max = atoi(argv[2]);
	int z_max = atoi(argv[3]);
	int tbx = atoi(argv[4]);
	int tby = atoi(argv[5]);
	int tbz = atoi(argv[6]);
	int c = atoi(argv[7]);
	// <--
	
	// allocate_grids -->
	u_0_0=((double * )malloc(((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double))));
	u_0_1=((double * )malloc(((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double))));
	a_1_0=((double * )malloc((((x_max*y_max)*z_max)*sizeof (double))));
	b_2_0=((double * )malloc((((x_max*y_max)*z_max)*sizeof (double))));
	c_3_0=((double * )malloc((((x_max*y_max)*z_max)*sizeof (double))));
	// <--
	
	
	// declare_GPU_grids -->
	double *  u_0_1_out_gpu;
	double *  u_0_0_gpu;
	double *  u_0_1_gpu;
	double *  a_1_0_gpu;
	double *  b_2_0_gpu;
	double *  c_3_0_gpu;
	dim3 thds(tbx, tby, tbz);
	dim3 blks((x_max/tbx), ((y_max*z_max)/(tby*tbz)), 1);
	// <--
	
	// allocate_GPU_grids -->
	hipMalloc(((void *  * )( & b_2_0_gpu)), (((x_max*y_max)*z_max)*sizeof (double)));
	hipMalloc(((void *  * )( & u_0_1_gpu)), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)));
	hipMalloc(((void *  * )( & u_0_1_out_gpu)), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double * )));
	hipMalloc(((void *  * )( & c_3_0_gpu)), (((x_max*y_max)*z_max)*sizeof (double)));
	hipMalloc(((void *  * )( & u_0_0_gpu)), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)));
	hipMalloc(((void *  * )( & a_1_0_gpu)), (((x_max*y_max)*z_max)*sizeof (double)));
	// <--
	
	// copy_grids_to_GPU -->
	hipMemcpy(((void * )b_2_0_gpu), ((void * )b_2_0), (((x_max*y_max)*z_max)*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )u_0_1_gpu), ((void * )u_0_1), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_3_0_gpu), ((void * )c_3_0), (((x_max*y_max)*z_max)*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )u_0_0_gpu), ((void * )u_0_0), ((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )a_1_0_gpu), ((void * )a_1_0), (((x_max*y_max)*z_max)*sizeof (double)), hipMemcpyHostToDevice);
	// <--
	
	
	// initialize_grids -->
	initialize<<<blks, thds>>>(u_0_0_gpu, u_0_1_gpu, a_1_0_gpu, b_2_0_gpu, c_3_0_gpu, x_max, y_max, z_max, tbx, tby, tbz, c);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Initialization]: %s.\n", hipGetErrorString (res));
	}
	
	long nFlopsPerStencil = 318;
	long nGridPointsCount = 5 * ((x_max*y_max)*z_max);
	long nBytesTransferred = 5 * (((((((x_max*y_max)*z_max)*sizeof (double))+(((x_max*y_max)*z_max)*sizeof (double)))+(((x_max*y_max)*z_max)*sizeof (double)))+((((x_max+6)*(y_max+6))*(z_max+6))*sizeof (double)))+(((x_max*y_max)*z_max)*sizeof (double)));
	
//	hipFuncSetCacheConfig(reinterpret_cast<const void*>(tricubic_interpolation), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(tricubic_interpolation), hipFuncCachePreferL1);
	
	// warm up
	// compute_stencil -->
	tricubic_interpolation<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, a_1_0_gpu, b_2_0_gpu, c_3_0_gpu, x_max, y_max, z_max, tbx, tby, tbz, c);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Stencil]: %s.\n", hipGetErrorString (res));
	}
	
	// run the benchmark
	tic ();
	for (i = 0; i < 5; i++)
	{
		// compute_stencil -->
		tricubic_interpolation<<<blks, thds>>>(( & u_0_1_out_gpu), u_0_0_gpu, u_0_1_gpu, a_1_0_gpu, b_2_0_gpu, c_3_0_gpu, x_max, y_max, z_max, tbx, tby, tbz, c);
		// <--
		
		hipDeviceSynchronize ();
	}
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);
	
	// free memory
	// deallocate_grids -->
	hipFree(((void * )b_2_0_gpu));
	hipFree(((void * )u_0_1_gpu));
	hipFree(((void * )u_0_1_out_gpu));
	hipFree(((void * )c_3_0_gpu));
	hipFree(((void * )u_0_0_gpu));
	hipFree(((void * )a_1_0_gpu));
	free(u_0_0);
	free(u_0_1);
	free(a_1_0);
	free(b_2_0);
	free(c_3_0);
	// <--
	
	
	hipDeviceReset ();
	return EXIT_SUCCESS;
}
