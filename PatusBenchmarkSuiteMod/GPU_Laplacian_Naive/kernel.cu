
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(u[0][0][0][1][0]=((((u[1][0][0][0][0]+(u[-1][0][0][0][0]+u[0][1][0][0][0]))+(u[0][-1][0][0][0]+(u[0][0][1][0][0]+u[0][0][-1][0][0])))*0.25)-u[0][0][0][0][0]))

*/
__global__ void laplacian(float *  *  u_0_1_out, float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max)
{
	//float *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+2) */
		_idx0=(((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+2);
		/* _idx1 = ((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x) */
		_idx1=(_idx0-2);
		/* _idx2 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+2)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+4)*t))+p_idx_x)+1) */
		_idx2=(((_idx1+x_max)+(2*t))+1);
		/* _idx3 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+(((((2*p_idx_z)+2)*t)+p_idx_y)*x_max))+(((4*p_idx_z)+4)*(t*t)))+((2*p_idx_y)*t))+p_idx_x)+1) */
		_idx3=(((_idx1-x_max)-(2*t))+1);
		/* _idx4 = (((((((((p_idx_z+2)*x_max)+(((2*p_idx_z)+4)*t))*y_max)+((((((2*p_idx_z)+4)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+8)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx4=((((_idx3+((x_max+(2*t))*y_max))+(((2*t)+1)*x_max))+(4*(t*t)))+(2*t));
		/* _idx5 = ((((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(((((2*p_idx_z)*t)+p_idx_y)+1)*x_max))+((4*p_idx_z)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx5=((((_idx1+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
		/* _idx6 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx6=(_idx1+1);
		u_0_1[_idx6]=((((u_0_0[_idx0]+(u_0_0[_idx1]+u_0_0[_idx2]))+(u_0_0[_idx3]+(u_0_0[_idx4]+u_0_0[_idx5])))*0.25)-u_0_0[_idx6]);
	}
}

__global__ void initialize(float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max)
{
	float *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = ((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x) */
		_idx0=((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x);
		u_0_0[_idx0]=0.1;
		/* _idx1 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+(((((2*p_idx_z)+2)*t)+p_idx_y)*x_max))+(((4*p_idx_z)+4)*(t*t)))+((2*p_idx_y)*t))+p_idx_x)+1) */
		_idx1=(((_idx0-x_max)-(2*t))+1);
		u_0_0[_idx1]=0.1;
		/* _idx2 = ((((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(((((2*p_idx_z)*t)+p_idx_y)+1)*x_max))+((4*p_idx_z)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx2=((((_idx0+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
		u_0_0[_idx2]=0.1;
		/* _idx3 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx3=(_idx0+1);
		u_0_0[_idx3]=0.1;
		/* _idx4 = (((((((((p_idx_z+2)*x_max)+(((2*p_idx_z)+4)*t))*y_max)+((((((2*p_idx_z)+4)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+8)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx4=(((_idx3+((x_max+(2*t))*y_max))+((2*t)*x_max))+(4*(t*t)));
		u_0_0[_idx4]=0.1;
		/* _idx5 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+2)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+4)*t))+p_idx_x)+1) */
		_idx5=((_idx3+x_max)+(2*t));
		u_0_0[_idx5]=0.1;
		/* _idx6 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+2) */
		_idx6=(_idx0+2);
		u_0_0[_idx6]=0.1;
		u_0_1[_idx3]=1.1;
	}
}

