
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(u[0][0][0][0][0]=((alpha*(ux[1][0][0][0][1]-ux[-1][0][0][0][1]))+((beta*(uy[0][1][0][0][2]-uy[0][-1][0][0][2]))+(gamma*(uz[0][0][1][0][3]-uz[0][0][-1][0][3])))))

*/
__global__ void divergence(float *  *  u_0_0_out, float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c)
{
/*
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
	*/
	
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int chunk_idx_x;
	int chunk_idx_x_max;
	int chunk_idx_y;
	int chunk_idx_y_max;
	int chunk_idx_z;
	int chunk_idx_z_max;
	int idx_1_2;
	int size_1_1;
	int size_1_2;
	//int t;
	int thd_idx_x;
	int thd_idx_y;
	int thd_idx_z;
	int thdblks_idx_x;
	int thdblks_idx_x_max;
	int thdblks_idx_y;
	int thdblks_idx_y_max;
	int thdblks_idx_z;
	int thdblks_idx_z_max;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	chunk_idx_x=(c*(threadIdx.x+(blockDim.x*blockIdx.x)));
	chunk_idx_x_max=(chunk_idx_x+c);
	chunk_idx_y=(threadIdx.y+(tmp*blockDim.y));
	chunk_idx_y_max=(chunk_idx_y+1);
	chunk_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	chunk_idx_z_max=(chunk_idx_z+1);
	thdblks_idx_x=(tbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	thdblks_idx_x_max=(thdblks_idx_x+tbx);
	thdblks_idx_y=(tby*(threadIdx.y+(tmp*blockDim.y)));
	thdblks_idx_y_max=(thdblks_idx_y+tby);
	thdblks_idx_z=(tbz*(threadIdx.z+(idx_1_2*blockDim.z)));
	thdblks_idx_z_max=(thdblks_idx_z+tbz);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in thdblks[t=t, s=(tbx, tby, tbz)][0] */
		/* Index bounds calculations for iterators in chunk[t=t, s=(c, 1, 1)][0] */
		/*
		for POINT thd[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in chunk[t=t, s=(:, :, :)][0] parallel 1 <level 2> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
			thd_idx_z=chunk_idx_z;
			thd_idx_y=chunk_idx_y;
			for (thd_idx_x=chunk_idx_x; thd_idx_x<(chunk_idx_x_max-0); thd_idx_x+=1)
			{
				/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
				/*
				u[t=(t+1), s=thd[t=?, s=?][0]][0]=stencil(u[t=t, s=thd[t=?, s=?][0]][0])
				*/
				/* _idx0 = (((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(thd_idx_y*x_max))+((2*t)*thd_idx_y))+thd_idx_x)+2) */
				_idx0=(((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(thd_idx_y*x_max))+((2*t)*thd_idx_y))+thd_idx_x)+2);
				/* _idx1 = ((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(thd_idx_y*x_max))+((2*t)*thd_idx_y))+thd_idx_x) */
				_idx1=(_idx0-2);
				/* _idx2 = ((((thd_idx_z*x_max)*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+2)*x_max))+thd_idx_x) */
				_idx2=(((_idx1-(((2*t)*thd_idx_z)*y_max))+((((2*t)*thd_idx_z)+2)*x_max))-((2*t)*thd_idx_y));
				/* _idx3 = ((((thd_idx_z*x_max)*y_max)+((((2*t)*thd_idx_z)+thd_idx_y)*x_max))+thd_idx_x) */
				_idx3=(_idx2-(2*x_max));
				/* _idx4 = (((((thd_idx_z+2)*x_max)*y_max)+(thd_idx_y*x_max))+thd_idx_x) */
				_idx4=((_idx3+((2*x_max)*y_max))-(((2*t)*thd_idx_z)*x_max));
				/* _idx5 = ((((thd_idx_z*x_max)*y_max)+(thd_idx_y*x_max))+thd_idx_x) */
				_idx5=(_idx4-((2*x_max)*y_max));
				u_0_0[_idx5]=((alpha*(ux_1_0[_idx0]-ux_1_0[_idx1]))+((beta*(uy_2_0[_idx2]-uy_2_0[_idx3]))+(gamma*(uz_3_0[_idx4]-uz_3_0[_idx5]))));
			}
		}
	}
}

__global__ void initialize(float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c)
{
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int chunk_idx_x;
	int chunk_idx_x_max;
	int chunk_idx_y;
	int chunk_idx_y_max;
	int chunk_idx_z;
	int chunk_idx_z_max;
	int idx_1_2;
	int size_1_1;
	int size_1_2;
	//int t;
	int thd_idx_x;
	int thd_idx_y;
	int thd_idx_z;
	int thdblks_idx_x;
	int thdblks_idx_x_max;
	int thdblks_idx_y;
	int thdblks_idx_y_max;
	int thdblks_idx_z;
	int thdblks_idx_z_max;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	chunk_idx_x=(c*(threadIdx.x+(blockDim.x*blockIdx.x)));
	chunk_idx_x_max=(chunk_idx_x+c);
	chunk_idx_y=(threadIdx.y+(tmp*blockDim.y));
	chunk_idx_y_max=(chunk_idx_y+1);
	chunk_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	chunk_idx_z_max=(chunk_idx_z+1);
	thdblks_idx_x=(tbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	thdblks_idx_x_max=(thdblks_idx_x+tbx);
	thdblks_idx_y=(tby*(threadIdx.y+(tmp*blockDim.y)));
	thdblks_idx_y_max=(thdblks_idx_y+tby);
	thdblks_idx_z=(tbz*(threadIdx.z+(idx_1_2*blockDim.z)));
	thdblks_idx_z_max=(thdblks_idx_z+tbz);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in thdblks[t=t, s=(tbx, tby, tbz)][0] */
		/* Index bounds calculations for iterators in chunk[t=t, s=(c, 1, 1)][0] */
		/*
		for POINT thd[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in chunk[t=t, s=(:, :, :)][0] parallel 1 <level 2> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
			thd_idx_z=chunk_idx_z;
			thd_idx_y=chunk_idx_y;
			for (thd_idx_x=chunk_idx_x; thd_idx_x<(chunk_idx_x_max-0); thd_idx_x+=1)
			{
				/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
				/*
				u[t=(t+1), s=thd[t=?, s=?][0]][0]=stencil(u[t=t, s=thd[t=?, s=?][0]][0])
				*/
				/* _idx0 = ((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(thd_idx_y*x_max))+((2*t)*thd_idx_y))+thd_idx_x) */
				_idx0=((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(thd_idx_y*x_max))+((2*t)*thd_idx_y))+thd_idx_x);
				u__ux_1[(t-1)][_idx0]=0.2;
				/* _idx1 = ((((thd_idx_z*x_max)*y_max)+((((2*t)*thd_idx_z)+thd_idx_y)*x_max))+thd_idx_x) */
				_idx1=(((_idx0-(((2*t)*thd_idx_z)*y_max))+(((2*t)*thd_idx_z)*x_max))-((2*t)*thd_idx_y));
				u__uy_2[(t-1)][_idx1]=0.30000000000000004;
				/* _idx2 = ((((thd_idx_z*x_max)*y_max)+(thd_idx_y*x_max))+thd_idx_x) */
				_idx2=(_idx1-(((2*t)*thd_idx_z)*x_max));
				u__uz_3[(t-1)][_idx2]=0.4;
				/* _idx3 = (((((thd_idx_z+2)*x_max)*y_max)+(thd_idx_y*x_max))+thd_idx_x) */
				_idx3=(_idx2+((2*x_max)*y_max));
				u__uz_3[(t-1)][_idx3]=0.4;
				/* _idx4 = ((((thd_idx_z*x_max)*y_max)+(((((2*t)*thd_idx_z)+thd_idx_y)+2)*x_max))+thd_idx_x) */
				_idx4=(_idx1+(2*x_max));
				u__uy_2[(t-1)][_idx4]=0.30000000000000004;
				/* _idx5 = (((((((thd_idx_z*x_max)+((2*t)*thd_idx_z))*y_max)+(thd_idx_y*x_max))+((2*t)*thd_idx_y))+thd_idx_x)+2) */
				_idx5=(_idx0+2);
				u__ux_1[(t-1)][_idx5]=0.2;
				u__u_0[(t-1)][_idx2]=0.1;
			}
		}
	}
}

