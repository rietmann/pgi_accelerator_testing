#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "patusrt.h"

// forward_decls -->
__global__ void initialize(float *  T_0_0, float *  T_0_1, float *  c_1_0, float *  c_2_0, float *  c_3_0, float *  c_4_0, float *  c_5_0, float *  c_6_0, float *  c_7_0, float *  c_8_0, float *  c_9_0, int x_max, int y_max, int z_max, int cbx);
__global__ void hyperthermia(float *  *  T_0_1_out, float *  T_0_0, float *  T_0_1, float *  c_1_0, float *  c_2_0, float *  c_3_0, float *  c_4_0, float *  c_5_0, float *  c_6_0, float *  c_7_0, float *  c_8_0, float *  c_9_0, int x_max, int y_max, int z_max, int cbx);

// <--


int main (int argc, char** argv)
{
	int i;
	hipError_t res;
	
	// prepare grids
	// declare_grids -->
	float *  T_0_1_out;
	float *  T_0_0;
	float *  T_0_1;
	float *  c_1_0;
	float *  c_2_0;
	float *  c_3_0;
	float *  c_4_0;
	float *  c_5_0;
	float *  c_6_0;
	float *  c_7_0;
	float *  c_8_0;
	float *  c_9_0;
	if ((argc!=5))
	{
		printf("Wrong number of parameters. Syntax:\n%s <x_max> <y_max> <z_max> <cbx>\n", argv[0]);
		exit(-1);
	}
	int x_max = atoi(argv[1]);
	int y_max = atoi(argv[2]);
	int z_max = atoi(argv[3]);
	int cbx = atoi(argv[4]);
	// <--
	
	// allocate_grids -->
	T_0_0=((float * )malloc(((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))));
	T_0_1=((float * )malloc(((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float))));
	c_1_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	c_2_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	c_3_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	c_4_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	c_5_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	c_6_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	c_7_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	c_8_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	c_9_0=((float * )malloc((((x_max*y_max)*z_max)*sizeof (float))));
	// <--
	
	
	// declare_GPU_grids -->
	float *  T_0_1_out_gpu;
	float *  T_0_0_gpu;
	float *  T_0_1_gpu;
	float *  c_1_0_gpu;
	float *  c_2_0_gpu;
	float *  c_3_0_gpu;
	float *  c_4_0_gpu;
	float *  c_5_0_gpu;
	float *  c_6_0_gpu;
	float *  c_7_0_gpu;
	float *  c_8_0_gpu;
	float *  c_9_0_gpu;
	dim3 thds(cbx, 1, 1);
	dim3 blks((x_max/cbx), (y_max*z_max), 1);
	// <--
	
	// allocate_GPU_grids -->
	hipMalloc(((void *  * )( & c_9_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & c_5_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & c_8_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & c_6_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & c_1_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & c_4_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & c_3_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & T_0_0_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)));
	hipMalloc(((void *  * )( & T_0_1_out_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float * )));
	hipMalloc(((void *  * )( & c_2_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & c_7_0_gpu)), (((x_max*y_max)*z_max)*sizeof (float)));
	hipMalloc(((void *  * )( & T_0_1_gpu)), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)));
	// <--
	
	// copy_grids_to_GPU -->
	hipMemcpy(((void * )c_9_0_gpu), ((void * )c_9_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_5_0_gpu), ((void * )c_5_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_8_0_gpu), ((void * )c_8_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_6_0_gpu), ((void * )c_6_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_1_0_gpu), ((void * )c_1_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_4_0_gpu), ((void * )c_4_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_3_0_gpu), ((void * )c_3_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )T_0_0_gpu), ((void * )T_0_0), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_2_0_gpu), ((void * )c_2_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )c_7_0_gpu), ((void * )c_7_0), (((x_max*y_max)*z_max)*sizeof (float)), hipMemcpyHostToDevice);
	hipMemcpy(((void * )T_0_1_gpu), ((void * )T_0_1), ((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)), hipMemcpyHostToDevice);
	// <--
	
	
	// initialize_grids -->
	initialize<<<blks, thds>>>(T_0_0_gpu, T_0_1_gpu, c_1_0_gpu, c_2_0_gpu, c_3_0_gpu, c_4_0_gpu, c_5_0_gpu, c_6_0_gpu, c_7_0_gpu, c_8_0_gpu, c_9_0_gpu, x_max, y_max, z_max, cbx);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Initialization]: %s.\n", hipGetErrorString (res));
	}
	
	long nFlopsPerStencil = 16;
	long nGridPointsCount = 5 * ((x_max*y_max)*z_max);
	long nBytesTransferred = 5 * (((((((((((((x_max*y_max)*z_max)*sizeof (float))+(((x_max*y_max)*z_max)*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float)))+((((x_max+4)*(y_max+4))*(z_max+4))*sizeof (float)))+(((x_max*y_max)*z_max)*sizeof (float)));
	
	// warm up
	// compute_stencil -->
	hyperthermia<<<blks, thds>>>(( & T_0_1_out_gpu), T_0_0_gpu, T_0_1_gpu, c_1_0_gpu, c_2_0_gpu, c_3_0_gpu, c_4_0_gpu, c_5_0_gpu, c_6_0_gpu, c_7_0_gpu, c_8_0_gpu, c_9_0_gpu, x_max, y_max, z_max, cbx);
	// <--
	
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Stencil]: %s.\n", hipGetErrorString (res));
	}
	
	// run the benchmark
	tic ();
	for (i = 0; i < 5; i++)
	{
		// compute_stencil -->
		hyperthermia<<<blks, thds>>>(( & T_0_1_out_gpu), T_0_0_gpu, T_0_1_gpu, c_1_0_gpu, c_2_0_gpu, c_3_0_gpu, c_4_0_gpu, c_5_0_gpu, c_6_0_gpu, c_7_0_gpu, c_8_0_gpu, c_9_0_gpu, x_max, y_max, z_max, cbx);
		// <--
		
		hipDeviceSynchronize ();
	}
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);
	
	// free memory
	// deallocate_grids -->
	hipFree(((void * )c_9_0_gpu));
	hipFree(((void * )c_5_0_gpu));
	hipFree(((void * )c_8_0_gpu));
	hipFree(((void * )c_6_0_gpu));
	hipFree(((void * )c_1_0_gpu));
	hipFree(((void * )c_4_0_gpu));
	hipFree(((void * )c_3_0_gpu));
	hipFree(((void * )T_0_0_gpu));
	hipFree(((void * )T_0_1_out_gpu));
	hipFree(((void * )c_2_0_gpu));
	hipFree(((void * )c_7_0_gpu));
	hipFree(((void * )T_0_1_gpu));
	free(T_0_0);
	free(T_0_1);
	free(c_1_0);
	free(c_2_0);
	free(c_3_0);
	free(c_4_0);
	free(c_5_0);
	free(c_6_0);
	free(c_7_0);
	free(c_8_0);
	free(c_9_0);
	// <--
	
	
	hipDeviceReset ();
	return EXIT_SUCCESS;
}
