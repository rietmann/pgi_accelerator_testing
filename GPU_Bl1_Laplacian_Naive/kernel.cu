
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(u[0][0][0][1][0]=((((u[1][0][0][0][0]+(u[-1][0][0][0][0]+u[0][1][0][0][0]))+(u[0][-1][0][0][0]+(u[0][0][1][0][0]+u[0][0][-1][0][0])))*0.25)-u[0][0][0][0][0]))

*/
__global__ void laplacian(float *  *  u_0_1_out, float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max, int cbx)
{
	//float *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int idx_1_2;
	int pt_idx_x;
	int pt_idx_y;
	int pt_idx_z;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	int v_idx_x;
	int v_idx_x_max;
	int v_idx_y;
	int v_idx_y_max;
	int v_idx_z;
	int v_idx_z_max;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	v_idx_x=(cbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	v_idx_x_max=(v_idx_x+cbx);
	v_idx_y=(threadIdx.y+(tmp*blockDim.y));
	v_idx_y_max=(v_idx_y+1);
	v_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	v_idx_z_max=(v_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in v[t=t, s=(cbx, 1, 1)][0] */
		/*
		for POINT pt[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in v[t=t, s=(:, :, :)][0] parallel 1 <level 1> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
			pt_idx_z=v_idx_z;
			pt_idx_y=v_idx_y;
			for (pt_idx_x=v_idx_x; pt_idx_x<(v_idx_x_max-0); pt_idx_x+=1)
			{
				/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
				/*
				v[t=(t+1), s=pt[t=?, s=?][0]][0]=stencil(v[t=t, s=pt[t=?, s=?][0]][0])
				*/
				/* _idx0 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+2) */
				_idx0=(((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+2);
				/* _idx1 = ((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x) */
				_idx1=(_idx0-2);
				/* _idx2 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+2)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+4)*t))+pt_idx_x)+1) */
				_idx2=(((_idx1+x_max)+(2*t))+1);
				/* _idx3 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+(((((2*pt_idx_z)+2)*t)+pt_idx_y)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+((2*pt_idx_y)*t))+pt_idx_x)+1) */
				_idx3=((_idx2-(2*x_max))-(4*t));
				/* _idx4 = (((((((((pt_idx_z+2)*x_max)+(((2*pt_idx_z)+4)*t))*y_max)+((((((2*pt_idx_z)+4)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+8)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx4=((((_idx2+((x_max+(2*t))*y_max))+(((2*t)-1)*x_max))+(4*(t*t)))-(2*t));
				/* _idx5 = ((((((((pt_idx_z*x_max)+((2*pt_idx_z)*t))*y_max)+(((((2*pt_idx_z)*t)+pt_idx_y)+1)*x_max))+((4*pt_idx_z)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx5=((((_idx1+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
				/* _idx6 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx6=(_idx1+1);
				u_0_1[_idx6]=((((u_0_0[_idx0]+(u_0_0[_idx1]+u_0_0[_idx2]))+(u_0_0[_idx3]+(u_0_0[_idx4]+u_0_0[_idx5])))*0.25)-u_0_0[_idx6]);
			}
		}
	}
}

__global__ void initialize(float *  u_0_0, float *  u_0_1, int x_max, int y_max, int z_max, int cbx)
{
	float *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int idx_1_2;
	int pt_idx_x;
	int pt_idx_y;
	int pt_idx_z;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	int v_idx_x;
	int v_idx_x_max;
	int v_idx_y;
	int v_idx_y_max;
	int v_idx_z;
	int v_idx_z_max;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	v_idx_x=(cbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	v_idx_x_max=(v_idx_x+cbx);
	v_idx_y=(threadIdx.y+(tmp*blockDim.y));
	v_idx_y_max=(v_idx_y+1);
	v_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	v_idx_z_max=(v_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in v[t=t, s=(cbx, 1, 1)][0] */
		/*
		for POINT pt[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in v[t=t, s=(:, :, :)][0] parallel 1 <level 1> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
			pt_idx_z=v_idx_z;
			pt_idx_y=v_idx_y;
			for (pt_idx_x=v_idx_x; pt_idx_x<(v_idx_x_max-0); pt_idx_x+=1)
			{
				/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
				/*
				v[t=(t+1), s=pt[t=?, s=?][0]][0]=stencil(v[t=t, s=pt[t=?, s=?][0]][0])
				*/
				/* _idx0 = ((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x) */
				_idx0=((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x);
				u_0_0[_idx0]=0.1;
				/* _idx1 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+(((((2*pt_idx_z)+2)*t)+pt_idx_y)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+((2*pt_idx_y)*t))+pt_idx_x)+1) */
				_idx1=(((_idx0-x_max)-(2*t))+1);
				u_0_0[_idx1]=0.1;
				/* _idx2 = ((((((((pt_idx_z*x_max)+((2*pt_idx_z)*t))*y_max)+(((((2*pt_idx_z)*t)+pt_idx_y)+1)*x_max))+((4*pt_idx_z)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx2=((((_idx0+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
				u_0_0[_idx2]=0.1;
				/* _idx3 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx3=(_idx0+1);
				u_0_0[_idx3]=0.1;
				/* _idx4 = (((((((((pt_idx_z+2)*x_max)+(((2*pt_idx_z)+4)*t))*y_max)+((((((2*pt_idx_z)+4)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+8)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx4=(((_idx3+((x_max+(2*t))*y_max))+((2*t)*x_max))+(4*(t*t)));
				u_0_0[_idx4]=0.1;
				/* _idx5 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+2)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+4)*t))+pt_idx_x)+1) */
				_idx5=((_idx3+x_max)+(2*t));
				u_0_0[_idx5]=0.1;
				/* _idx6 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+2) */
				_idx6=(_idx0+2);
				u_0_0[_idx6]=0.1;
				u_0_1[_idx3]=1.1;
			}
		}
	}
}

