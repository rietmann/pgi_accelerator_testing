
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(u[0][0][0][1][0]=(a*((((u[-3][0][0][0][0]+(u[0][-3][0][0][0]+u[0][0][-3][0][0]))*-2.0)+(((u[-2][0][0][0][0]+(u[0][-2][0][0][0]+u[0][0][-2][0][0]))*15.0)+((u[-1][0][0][0][0]+(u[0][-1][0][0][0]+u[0][0][-1][0][0]))*-60.0)))+((u[0][0][0][0][0]*20.0)+(((u[1][0][0][0][0]+(u[0][1][0][0][0]+u[0][0][1][0][0]))*30.0)+((u[2][0][0][0][0]+(u[0][2][0][0][0]+u[0][0][2][0][0]))*-3.0))))))

*/
__global__ void upstream_5_3d(double *  *  u_0_1_out, double *  u_0_0, double *  u_0_1, double a, int x_max, int y_max, int z_max, int cbx)
{
//	double *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx10;
	int _idx11;
	int _idx12;
	int _idx13;
	int _idx14;
	int _idx15;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int _idx8;
	int _idx9;
	int idx_1_2;
	int pt_idx_x;
	int pt_idx_y;
	int pt_idx_z;
	int size_1_1;
	int size_1_2;
//	int t;
	int tmp;
	int v_idx_x;
	int v_idx_x_max;
	int v_idx_y;
	int v_idx_y_max;
	int v_idx_z;
	int v_idx_z_max;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	v_idx_x=(cbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	v_idx_x_max=(v_idx_x+cbx);
	v_idx_y=(threadIdx.y+(tmp*blockDim.y));
	v_idx_y_max=(v_idx_y+1);
	v_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	v_idx_z_max=(v_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
//	for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in v[t=t, s=(cbx, 1, 1)][0] */
		/*
		for POINT pt[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in v[t=t, s=(:, :, :)][0] parallel 1 <level 1> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
			pt_idx_z=v_idx_z;
			pt_idx_y=v_idx_y;
			for (pt_idx_x=v_idx_x; pt_idx_x<(v_idx_x_max-0); pt_idx_x+=1)
			{
				/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
				/*
				v[t=(t+1), s=pt[t=?, s=?][0]][0]=stencil(v[t=t, s=pt[t=?, s=?][0]][0])
				*/
				/* _idx0 = ((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x) */
				_idx0=((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x);
				/* _idx1 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+(((((5*pt_idx_z)+15)*t)+pt_idx_y)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+((5*pt_idx_y)*t))+pt_idx_x)+3) */
				_idx1=(((_idx0-(3*x_max))-(15*t))+3);
				/* _idx2 = ((((((((pt_idx_z*x_max)+((5*pt_idx_z)*t))*y_max)+(((((5*pt_idx_z)*t)+pt_idx_y)+3)*x_max))+((25*pt_idx_z)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx2=((((_idx1+(((-3*x_max)-(15*t))*y_max))+((3-(15*t))*x_max))-(75*(t*t)))+(15*t));
				/* _idx3 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+1) */
				_idx3=(_idx0+1);
				/* _idx4 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+1)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+5)*t))+pt_idx_x)+3) */
				_idx4=((_idx1+x_max)+(5*t));
				/* _idx5 = (((((((((pt_idx_z+1)*x_max)+(((5*pt_idx_z)+5)*t))*y_max)+((((((5*pt_idx_z)+5)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+25)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx5=(((_idx2+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				/* _idx6 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+2) */
				_idx6=(_idx0+2);
				/* _idx7 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+2)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+10)*t))+pt_idx_x)+3) */
				_idx7=((_idx4+x_max)+(5*t));
				/* _idx8 = (((((((((pt_idx_z+2)*x_max)+(((5*pt_idx_z)+10)*t))*y_max)+((((((5*pt_idx_z)+10)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+50)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx8=(((_idx5+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				/* _idx9 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx9=(_idx0+3);
				/* _idx10 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+4) */
				_idx10=(_idx0+4);
				/* _idx11 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+4)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+20)*t))+pt_idx_x)+3) */
				_idx11=((_idx9+x_max)+(5*t));
				/* _idx12 = (((((((((pt_idx_z+4)*x_max)+(((5*pt_idx_z)+20)*t))*y_max)+((((((5*pt_idx_z)+20)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+100)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx12=(((_idx9+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				/* _idx13 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+5) */
				_idx13=(_idx0+5);
				/* _idx14 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+5)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+25)*t))+pt_idx_x)+3) */
				_idx14=((_idx11+x_max)+(5*t));
				/* _idx15 = (((((((((pt_idx_z+5)*x_max)+(((5*pt_idx_z)+25)*t))*y_max)+((((((5*pt_idx_z)+25)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+125)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx15=(((_idx12+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u_0_1[_idx9]=(a*((((u_0_0[_idx0]+(u_0_0[_idx1]+u_0_0[_idx2]))*-2.0)+(((u_0_0[_idx3]+(u_0_0[_idx4]+u_0_0[_idx5]))*15.0)+((u_0_0[_idx6]+(u_0_0[_idx7]+u_0_0[_idx8]))*-60.0)))+((u_0_0[_idx9]*20.0)+(((u_0_0[_idx10]+(u_0_0[_idx11]+u_0_0[_idx12]))*30.0)+((u_0_0[_idx13]+(u_0_0[_idx14]+u_0_0[_idx15]))*-3.0)))));
			}
		}
	}
}

__global__ void initialize(double *  u_0_0, double *  u_0_1, double a, int x_max, int y_max, int z_max, int cbx)
{
	double *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx10;
	int _idx11;
	int _idx12;
	int _idx13;
	int _idx14;
	int _idx15;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int _idx8;
	int _idx9;
	int idx_1_2;
	int pt_idx_x;
	int pt_idx_y;
	int pt_idx_z;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	int v_idx_x;
	int v_idx_x_max;
	int v_idx_y;
	int v_idx_y_max;
	int v_idx_z;
	int v_idx_z_max;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	v_idx_x=(cbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	v_idx_x_max=(v_idx_x+cbx);
	v_idx_y=(threadIdx.y+(tmp*blockDim.y));
	v_idx_y_max=(v_idx_y+1);
	v_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	v_idx_z_max=(v_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in v[t=t, s=(cbx, 1, 1)][0] */
		/*
		for POINT pt[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in v[t=t, s=(:, :, :)][0] parallel 1 <level 1> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
			pt_idx_z=v_idx_z;
			pt_idx_y=v_idx_y;
			for (pt_idx_x=v_idx_x; pt_idx_x<(v_idx_x_max-0); pt_idx_x+=1)
			{
				/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
				/*
				v[t=(t+1), s=pt[t=?, s=?][0]][0]=stencil(v[t=t, s=pt[t=?, s=?][0]][0])
				*/
				/* _idx0 = ((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x) */
				_idx0=((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x);
				u_0_0[_idx0]=0.1;
				/* _idx1 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+1) */
				_idx1=(_idx0+1);
				u_0_0[_idx1]=0.1;
				/* _idx2 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+2) */
				_idx2=(_idx0+2);
				u_0_0[_idx2]=0.1;
				/* _idx3 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+(((((5*pt_idx_z)+15)*t)+pt_idx_y)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+((5*pt_idx_y)*t))+pt_idx_x)+3) */
				_idx3=(((_idx0-(3*x_max))-(15*t))+3);
				u_0_0[_idx3]=0.1;
				/* _idx4 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+1)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+5)*t))+pt_idx_x)+3) */
				_idx4=((_idx3+x_max)+(5*t));
				u_0_0[_idx4]=0.1;
				/* _idx5 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+2)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+10)*t))+pt_idx_x)+3) */
				_idx5=((_idx4+x_max)+(5*t));
				u_0_0[_idx5]=0.1;
				/* _idx6 = ((((((((pt_idx_z*x_max)+((5*pt_idx_z)*t))*y_max)+(((((5*pt_idx_z)*t)+pt_idx_y)+3)*x_max))+((25*pt_idx_z)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx6=((((_idx1+(((-3*x_max)-(15*t))*y_max))-((15*t)*x_max))-(75*(t*t)))+2);
				u_0_0[_idx6]=0.1;
				/* _idx7 = (((((((((pt_idx_z+1)*x_max)+(((5*pt_idx_z)+5)*t))*y_max)+((((((5*pt_idx_z)+5)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+25)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx7=(((_idx6+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u_0_0[_idx7]=0.1;
				/* _idx8 = (((((((((pt_idx_z+2)*x_max)+(((5*pt_idx_z)+10)*t))*y_max)+((((((5*pt_idx_z)+10)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+50)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx8=(((_idx7+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u_0_0[_idx8]=0.1;
				/* _idx9 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx9=(_idx0+3);
				u_0_0[_idx9]=0.1;
				/* _idx10 = (((((((((pt_idx_z+4)*x_max)+(((5*pt_idx_z)+20)*t))*y_max)+((((((5*pt_idx_z)+20)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+100)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx10=(((_idx9+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u_0_0[_idx10]=0.1;
				/* _idx11 = (((((((((pt_idx_z+5)*x_max)+(((5*pt_idx_z)+25)*t))*y_max)+((((((5*pt_idx_z)+25)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+125)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+3) */
				_idx11=(((_idx10+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u_0_0[_idx11]=0.1;
				/* _idx12 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+4)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+20)*t))+pt_idx_x)+3) */
				_idx12=((_idx9+x_max)+(5*t));
				u_0_0[_idx12]=0.1;
				/* _idx13 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+5)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+25)*t))+pt_idx_x)+3) */
				_idx13=((_idx12+x_max)+(5*t));
				u_0_0[_idx13]=0.1;
				/* _idx14 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+4) */
				_idx14=(_idx0+4);
				u_0_0[_idx14]=0.1;
				/* _idx15 = (((((((((pt_idx_z+3)*x_max)+(((5*pt_idx_z)+15)*t))*y_max)+((((((5*pt_idx_z)+15)*t)+pt_idx_y)+3)*x_max))+(((25*pt_idx_z)+75)*(t*t)))+(((5*pt_idx_y)+15)*t))+pt_idx_x)+5) */
				_idx15=(_idx0+5);
				u_0_0[_idx15]=0.1;
				u_0_1[_idx9]=1.1;
			}
		}
	}
}

