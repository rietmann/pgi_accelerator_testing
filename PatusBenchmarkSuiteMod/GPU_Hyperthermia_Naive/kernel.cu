
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1
/*
(T[0][0][0][1][0]=((((T[0][0][0][0][0]*((c[0][0][0][0][1]*T[0][0][0][0][0])+c[0][0][0][0][2]))+c[0][0][0][0][3])+((c[0][0][0][0][4]*T[-1][0][0][0][0])+(c[0][0][0][0][5]*T[1][0][0][0][0])))+(((c[0][0][0][0][6]*T[0][-1][0][0][0])+(c[0][0][0][0][7]*T[0][1][0][0][0]))+((c[0][0][0][0][8]*T[0][0][-1][0][0])+(c[0][0][0][0][9]*T[0][0][1][0][0])))))

*/
__global__ void hyperthermia(float *  *  T_0_1_out, float *  T_0_0, float *  T_0_1, float *  c_1_0, float *  c_2_0, float *  c_3_0, float *  c_4_0, float *  c_5_0, float *  c_6_0, float *  c_7_0, float *  c_8_0, float *  c_9_0, int x_max, int y_max, int z_max)
{
/*
	const float *  const u__c_1[16] =  { c_1_0 } ;
	const float *  const u__c_2[16] =  { c_2_0 } ;
	const float *  const u__c_3[16] =  { c_3_0 } ;
	const float *  const u__c_4[16] =  { c_4_0 } ;
	const float *  const u__c_5[16] =  { c_5_0 } ;
	const float *  const u__c_6[16] =  { c_6_0 } ;
	const float *  const u__c_7[16] =  { c_7_0 } ;
	const float *  const u__c_8[16] =  { c_8_0 } ;
	const float *  const u__c_9[16] =  { c_9_0 } ;
	float *  const u__T_0[16] =  { T_0_0, T_0_1 } ;
*/
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx0=(((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1);
		/* _idx1 = ((((p_idx_z*x_max)*y_max)+(p_idx_y*x_max))+p_idx_x) */
		_idx1=(((((_idx0+(((((-2*p_idx_z)-2)*t)-x_max)*y_max))+(((((-2*p_idx_z)-2)*t)-1)*x_max))+(((-4*p_idx_z)-4)*(t*t)))+(((-2*p_idx_y)-2)*t))-1);
		/* _idx2 = ((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x) */
		_idx2=(_idx0-1);
		/* _idx3 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+2) */
		_idx3=(_idx2+2);
		/* _idx4 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+(((((2*p_idx_z)+2)*t)+p_idx_y)*x_max))+(((4*p_idx_z)+4)*(t*t)))+((2*p_idx_y)*t))+p_idx_x)+1) */
		_idx4=((_idx0-x_max)-(2*t));
		/* _idx5 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+2)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+4)*t))+p_idx_x)+1) */
		_idx5=((_idx0+x_max)+(2*t));
		/* _idx6 = ((((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(((((2*p_idx_z)*t)+p_idx_y)+1)*x_max))+((4*p_idx_z)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx6=((((_idx2+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
		/* _idx7 = (((((((((p_idx_z+2)*x_max)+(((2*p_idx_z)+4)*t))*y_max)+((((((2*p_idx_z)+4)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+8)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx7=(((_idx0+((x_max+(2*t))*y_max))+((2*t)*x_max))+(4*(t*t)));
		T_0_1[_idx0]=((((T_0_0[_idx0]*((c_1_0[_idx1]*T_0_0[_idx0])+c_2_0[_idx1]))+c_3_0[_idx1])+((c_4_0[_idx1]*T_0_0[_idx2])+(c_5_0[_idx1]*T_0_0[_idx3])))+(((c_6_0[_idx1]*T_0_0[_idx4])+(c_7_0[_idx1]*T_0_0[_idx5]))+((c_8_0[_idx1]*T_0_0[_idx6])+(c_9_0[_idx1]*T_0_0[_idx7]))));
	}
}

__global__ void initialize(float *  T_0_0, float *  T_0_1, float *  c_1_0, float *  c_2_0, float *  c_3_0, float *  c_4_0, float *  c_5_0, float *  c_6_0, float *  c_7_0, float *  c_8_0, float *  c_9_0, int x_max, int y_max, int z_max)
{
	 float *  const u__c_1[16] =  { c_1_0 } ;
	 float *  const u__c_2[16] =  { c_2_0 } ;
	 float *  const u__c_3[16] =  { c_3_0 } ;
	 float *  const u__c_4[16] =  { c_4_0 } ;
	 float *  const u__c_5[16] =  { c_5_0 } ;
	 float *  const u__c_6[16] =  { c_6_0 } ;
	 float *  const u__c_7[16] =  { c_7_0 } ;
	 float *  const u__c_8[16] =  { c_8_0 } ;
	 float *  const u__c_9[16] =  { c_9_0 } ;
	float *  const u__T_0[16] =  { T_0_0, T_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int idx_1_2;
	int p_idx_x;
	int p_idx_x_max;
	int p_idx_y;
	int p_idx_y_max;
	int p_idx_z;
	int p_idx_z_max;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	p_idx_x=(threadIdx.x+(blockDim.x*blockIdx.x));
	p_idx_x_max=(p_idx_x+1);
	p_idx_y=(threadIdx.y+(tmp*blockDim.y));
	p_idx_y_max=(p_idx_y+1);
	p_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	p_idx_z_max=(p_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in p[t=t, s=(1, 1, 1)][0] */
		/*
		u[t=(t+1), s=p[t=?, s=?][0]][0]=stencil(u[t=t, s=p[t=?, s=?][0]][0])
		*/
		/* _idx0 = ((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x) */
		_idx0=((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x);
		u__T_0[(t-1)][_idx0]=0.1;
		/* _idx1 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+(((((2*p_idx_z)+2)*t)+p_idx_y)*x_max))+(((4*p_idx_z)+4)*(t*t)))+((2*p_idx_y)*t))+p_idx_x)+1) */
		_idx1=(((_idx0-x_max)-(2*t))+1);
		u__T_0[(t-1)][_idx1]=0.1;
		/* _idx2 = ((((((((p_idx_z*x_max)+((2*p_idx_z)*t))*y_max)+(((((2*p_idx_z)*t)+p_idx_y)+1)*x_max))+((4*p_idx_z)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx2=((((_idx0+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
		u__T_0[(t-1)][_idx2]=0.1;
		/* _idx3 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx3=(_idx0+1);
		u__T_0[(t-1)][_idx3]=0.1;
		/* _idx4 = ((((p_idx_z*x_max)*y_max)+(p_idx_y*x_max))+p_idx_x) */
		_idx4=(((((_idx2-(((2*p_idx_z)*t)*y_max))+((((-2*p_idx_z)*t)-1)*x_max))-((4*p_idx_z)*(t*t)))+(((-2*p_idx_y)-2)*t))-1);
		u__c_1[(t-1)][_idx4]=0.2;
		u__c_2[(t-1)][_idx4]=0.30000000000000004;
		u__c_3[(t-1)][_idx4]=0.4;
		u__c_4[(t-1)][_idx4]=0.5;
		u__c_5[(t-1)][_idx4]=0.6000000000000001;
		u__c_6[(t-1)][_idx4]=0.7000000000000001;
		u__c_7[(t-1)][_idx4]=0.8;
		u__c_8[(t-1)][_idx4]=0.9;
		u__c_9[(t-1)][_idx4]=1.0;
		/* _idx5 = (((((((((p_idx_z+2)*x_max)+(((2*p_idx_z)+4)*t))*y_max)+((((((2*p_idx_z)+4)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+8)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+1) */
		_idx5=(((_idx3+((x_max+(2*t))*y_max))+((2*t)*x_max))+(4*(t*t)));
		u__T_0[(t-1)][_idx5]=0.1;
		/* _idx6 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+2)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+4)*t))+p_idx_x)+1) */
		_idx6=((_idx3+x_max)+(2*t));
		u__T_0[(t-1)][_idx6]=0.1;
		/* _idx7 = (((((((((p_idx_z+1)*x_max)+(((2*p_idx_z)+2)*t))*y_max)+((((((2*p_idx_z)+2)*t)+p_idx_y)+1)*x_max))+(((4*p_idx_z)+4)*(t*t)))+(((2*p_idx_y)+2)*t))+p_idx_x)+2) */
		_idx7=(_idx0+2);
		u__T_0[(t-1)][_idx7]=0.1;
		u__T_0[t][_idx3]=1.1;
	}
}

