
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(ux[0][0][0][0][1]=(alpha*(u[1][0][0][0][0]+u[-1][0][0][0][0])))
(uy[0][0][0][0][2]=(beta*(u[0][1][0][0][0]+u[0][-1][0][0][0])))
(uz[0][0][0][0][3]=(gamma*(u[0][0][1][0][0]+u[0][0][-1][0][0])))

*/
__global__ void gradient(float *  *  ux_1_0_out, float *  *  uy_2_0_out, float *  *  uz_3_0_out, float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max, int cbx)
{
/*
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
*/

	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int idx_1_2;
	int pt_idx_x;
	int pt_idx_y;
	int pt_idx_z;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	int v_idx_x;
	int v_idx_x_max;
	int v_idx_y;
	int v_idx_y_max;
	int v_idx_z;
	int v_idx_z_max;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	v_idx_x=(cbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	v_idx_x_max=(v_idx_x+cbx);
	v_idx_y=(threadIdx.y+(tmp*blockDim.y));
	v_idx_y_max=(v_idx_y+1);
	v_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	v_idx_z_max=(v_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in v[t=t, s=(cbx, 1, 1)][0] */
		/*
		for POINT pt[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in v[t=t, s=(:, :, :)][0] parallel 1 <level 1> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
			pt_idx_z=v_idx_z;
			pt_idx_y=v_idx_y;
			for (pt_idx_x=v_idx_x; pt_idx_x<(v_idx_x_max-0); pt_idx_x+=1)
			{
				/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
				/*
				v[t=(t+1), s=pt[t=?, s=?][0]][0]=stencil(v[t=t, s=pt[t=?, s=?][0]][0])
				*/
				/* _idx0 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+2) */
				_idx0=(((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+2);
				/* _idx1 = ((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x) */
				_idx1=(_idx0-2);
				/* _idx2 = ((((pt_idx_z*x_max)*y_max)+(pt_idx_y*x_max))+pt_idx_x) */
				_idx2=((((_idx1+(((((-2*pt_idx_z)-2)*t)-x_max)*y_max))+(((((-2*pt_idx_z)-2)*t)-1)*x_max))+(((-4*pt_idx_z)-4)*(t*t)))+(((-2*pt_idx_y)-2)*t));
				ux_1_0[_idx2]=(alpha*(u_0_0[_idx0]+u_0_0[_idx1]));
				/* _idx3 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+2)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+4)*t))+pt_idx_x)+1) */
				_idx3=(((_idx1+x_max)+(2*t))+1);
				/* _idx4 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+(((((2*pt_idx_z)+2)*t)+pt_idx_y)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+((2*pt_idx_y)*t))+pt_idx_x)+1) */
				_idx4=((_idx3-(2*x_max))-(4*t));
				uy_2_0[_idx2]=(beta*(u_0_0[_idx3]+u_0_0[_idx4]));
				/* _idx5 = (((((((((pt_idx_z+2)*x_max)+(((2*pt_idx_z)+4)*t))*y_max)+((((((2*pt_idx_z)+4)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+8)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx5=((((_idx3+((x_max+(2*t))*y_max))+(((2*t)-1)*x_max))+(4*(t*t)))-(2*t));
				/* _idx6 = ((((((((pt_idx_z*x_max)+((2*pt_idx_z)*t))*y_max)+(((((2*pt_idx_z)*t)+pt_idx_y)+1)*x_max))+((4*pt_idx_z)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx6=((((_idx1+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
				uz_3_0[_idx2]=(gamma*(u_0_0[_idx5]+u_0_0[_idx6]));
			}
		}
	}
}

__global__ void initialize(float *  u_0_0, float *  ux_1_0, float *  uy_2_0, float *  uz_3_0, float alpha, float beta, float gamma, int x_max, int y_max, int z_max, int cbx)
{
	float *  const u__u_0[16] =  { u_0_0 } ;
	float *  const u__ux_1[16] =  { ux_1_0 } ;
	float *  const u__uy_2[16] =  { uy_2_0 } ;
	float *  const u__uz_3[16] =  { uz_3_0 } ;
	int _idx0;
	int _idx1;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int idx_1_2;
	int pt_idx_x;
	int pt_idx_y;
	int pt_idx_z;
	int size_1_1;
	int size_1_2;
	//int t;
	int tmp;
	int v_idx_x;
	int v_idx_x_max;
	int v_idx_y;
	int v_idx_y_max;
	int v_idx_z;
	int v_idx_z_max;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	v_idx_x=(cbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	v_idx_x_max=(v_idx_x+cbx);
	v_idx_y=(threadIdx.y+(tmp*blockDim.y));
	v_idx_y_max=(v_idx_y+1);
	v_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	v_idx_z_max=(v_idx_z+1);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in v[t=t, s=(cbx, 1, 1)][0] */
		/*
		for POINT pt[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in v[t=t, s=(:, :, :)][0] parallel 1 <level 1> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
			pt_idx_z=v_idx_z;
			pt_idx_y=v_idx_y;
			for (pt_idx_x=v_idx_x; pt_idx_x<(v_idx_x_max-0); pt_idx_x+=1)
			{
				/* Index bounds calculations for iterators in pt[t=t, s=(1, 1, 1)][0] */
				/*
				v[t=(t+1), s=pt[t=?, s=?][0]][0]=stencil(v[t=t, s=pt[t=?, s=?][0]][0])
				*/
				/* _idx0 = ((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x) */
				_idx0=((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x);
				u_0_0[_idx0]=0.1;
				/* _idx1 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+2) */
				_idx1=(_idx0+2);
				u_0_0[_idx1]=0.1;
				/* _idx2 = ((((pt_idx_z*x_max)*y_max)+(pt_idx_y*x_max))+pt_idx_x) */
				_idx2=((((_idx0+(((((-2*pt_idx_z)-2)*t)-x_max)*y_max))+(((((-2*pt_idx_z)-2)*t)-1)*x_max))+(((-4*pt_idx_z)-4)*(t*t)))+(((-2*pt_idx_y)-2)*t));
				ux_1_0[_idx2]=0.2;
				/* _idx3 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+(((((2*pt_idx_z)+2)*t)+pt_idx_y)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+((2*pt_idx_y)*t))+pt_idx_x)+1) */
				_idx3=(((_idx0-x_max)-(2*t))+1);
				u_0_0[_idx3]=0.1;
				/* _idx4 = (((((((((pt_idx_z+1)*x_max)+(((2*pt_idx_z)+2)*t))*y_max)+((((((2*pt_idx_z)+2)*t)+pt_idx_y)+2)*x_max))+(((4*pt_idx_z)+4)*(t*t)))+(((2*pt_idx_y)+4)*t))+pt_idx_x)+1) */
				_idx4=((_idx3+(2*x_max))+(4*t));
				u_0_0[_idx4]=0.1;
				uy_2_0[_idx2]=0.30000000000000004;
				/* _idx5 = ((((((((pt_idx_z*x_max)+((2*pt_idx_z)*t))*y_max)+(((((2*pt_idx_z)*t)+pt_idx_y)+1)*x_max))+((4*pt_idx_z)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx5=((((_idx0+((( - x_max)-(2*t))*y_max))-((2*t)*x_max))-(4*(t*t)))+1);
				u_0_0[_idx5]=0.1;
				/* _idx6 = (((((((((pt_idx_z+2)*x_max)+(((2*pt_idx_z)+4)*t))*y_max)+((((((2*pt_idx_z)+4)*t)+pt_idx_y)+1)*x_max))+(((4*pt_idx_z)+8)*(t*t)))+(((2*pt_idx_y)+2)*t))+pt_idx_x)+1) */
				_idx6=((((_idx4+((x_max+(2*t))*y_max))+(((2*t)-1)*x_max))+(4*(t*t)))-(2*t));
				u_0_0[_idx6]=0.1;
				uz_3_0[_idx2]=0.4;
			}
		}
	}
}

