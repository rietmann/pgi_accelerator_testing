
#include <hip/hip_runtime.h>
#define t_max 1
#define t 1

/*
(u[0][0][0][1][0]=(a*((((u[-3][0][0][0][0]+(u[0][-3][0][0][0]+u[0][0][-3][0][0]))*-2.0)+(((u[-2][0][0][0][0]+(u[0][-2][0][0][0]+u[0][0][-2][0][0]))*15.0)+((u[-1][0][0][0][0]+(u[0][-1][0][0][0]+u[0][0][-1][0][0]))*-60.0)))+((u[0][0][0][0][0]*20.0)+(((u[1][0][0][0][0]+(u[0][1][0][0][0]+u[0][0][1][0][0]))*30.0)+((u[2][0][0][0][0]+(u[0][2][0][0][0]+u[0][0][2][0][0]))*-3.0))))))

*/
__global__ void upstream_5_3d(double *  *  u_0_1_out, double *  u_0_0, double *  u_0_1, double a, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c)
{
	//double *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx10;
	int _idx11;
	int _idx12;
	int _idx13;
	int _idx14;
	int _idx15;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int _idx8;
	int _idx9;
	int chunk_idx_x;
	int chunk_idx_x_max;
	int chunk_idx_y;
	int chunk_idx_y_max;
	int chunk_idx_z;
	int chunk_idx_z_max;
	int idx_1_2;
	int size_1_1;
	int size_1_2;
	//int t;
	int thd_idx_x;
	int thd_idx_y;
	int thd_idx_z;
	int thdblks_idx_x;
	int thdblks_idx_x_max;
	int thdblks_idx_y;
	int thdblks_idx_y_max;
	int thdblks_idx_z;
	int thdblks_idx_z_max;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	chunk_idx_x=(c*(threadIdx.x+(blockDim.x*blockIdx.x)));
	chunk_idx_x_max=(chunk_idx_x+c);
	chunk_idx_y=(threadIdx.y+(tmp*blockDim.y));
	chunk_idx_y_max=(chunk_idx_y+1);
	chunk_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	chunk_idx_z_max=(chunk_idx_z+1);
	thdblks_idx_x=(tbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	thdblks_idx_x_max=(thdblks_idx_x+tbx);
	thdblks_idx_y=(tby*(threadIdx.y+(tmp*blockDim.y)));
	thdblks_idx_y_max=(thdblks_idx_y+tby);
	thdblks_idx_z=(tbz*(threadIdx.z+(idx_1_2*blockDim.z)));
	thdblks_idx_z_max=(thdblks_idx_z+tbz);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in thdblks[t=t, s=(tbx, tby, tbz)][0] */
		/* Index bounds calculations for iterators in chunk[t=t, s=(c, 1, 1)][0] */
		/*
		for POINT thd[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in chunk[t=t, s=(:, :, :)][0] parallel 1 <level 2> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
			thd_idx_z=chunk_idx_z;
			thd_idx_y=chunk_idx_y;
			for (thd_idx_x=chunk_idx_x; thd_idx_x<(chunk_idx_x_max-0); thd_idx_x+=1)
			{
				/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
				/*
				u[t=(t+1), s=thd[t=?, s=?][0]][0]=stencil(u[t=t, s=thd[t=?, s=?][0]][0])
				*/
				/* _idx0 = (((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t)) */
				_idx0=(((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t));
				/* _idx1 = (((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+(((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+3) */
				_idx1=(((_idx0-(3*x_max))-(15*t))+3);
				/* _idx2 = (((((((((thd_idx_z*x_max)+((5*t)*thd_idx_z))*y_max)+(((((5*t)*thd_idx_z)+thd_idx_y)+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(15*t))+3) */
				_idx2=((((_idx0+(((-3*x_max)-(15*t))*y_max))-((15*t)*x_max))-(75*(t*t)))+3);
				/* _idx3 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+1) */
				_idx3=(_idx0+1);
				/* _idx4 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+1)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(5*t))+3) */
				_idx4=((_idx1+x_max)+(5*t));
				/* _idx5 = ((((((((((((thd_idx_z+1)*x_max)+((5*t)*thd_idx_z))+(5*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(5*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(25*(t*t)))+(15*t))+3) */
				_idx5=(((_idx2+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				/* _idx6 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+2) */
				_idx6=(_idx3+1);
				/* _idx7 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+2)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(10*t))+3) */
				_idx7=((_idx4+x_max)+(5*t));
				/* _idx8 = ((((((((((((thd_idx_z+2)*x_max)+((5*t)*thd_idx_z))+(10*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(10*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(50*(t*t)))+(15*t))+3) */
				_idx8=(((_idx5+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				/* _idx9 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+3) */
				_idx9=(_idx3+2);
				/* _idx10 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+4) */
				_idx10=(_idx3+3);
				/* _idx11 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+4)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(20*t))+3) */
				_idx11=((_idx9+x_max)+(5*t));
				/* _idx12 = ((((((((((((thd_idx_z+4)*x_max)+((5*t)*thd_idx_z))+(20*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(20*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(100*(t*t)))+(15*t))+3) */
				_idx12=(((_idx9+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				/* _idx13 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+5) */
				_idx13=(_idx3+4);
				/* _idx14 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+5)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(25*t))+3) */
				_idx14=((_idx11+x_max)+(5*t));
				/* _idx15 = ((((((((((((thd_idx_z+5)*x_max)+((5*t)*thd_idx_z))+(25*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(25*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(125*(t*t)))+(15*t))+3) */
				_idx15=(((_idx12+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u_0_1[_idx9]=(a*((((u_0_0[_idx0]+(u_0_0[_idx1]+u_0_0[_idx2]))*-2.0)+(((u_0_0[_idx3]+(u_0_0[_idx4]+u_0_0[_idx5]))*15.0)+((u_0_0[_idx6]+(u_0_0[_idx7]+u_0_0[_idx8]))*-60.0)))+((u_0_0[_idx9]*20.0)+(((u_0_0[_idx10]+(u_0_0[_idx11]+u_0_0[_idx12]))*30.0)+((u_0_0[_idx13]+(u_0_0[_idx14]+u_0_0[_idx15]))*-3.0)))));
			}
		}
	}
}

__global__ void initialize(double *  u_0_0, double *  u_0_1, double a, int x_max, int y_max, int z_max, int tbx, int tby, int tbz, int c)
{
	double *  const u__u_0[16] =  { u_0_0, u_0_1 } ;
	int _idx0;
	int _idx1;
	int _idx10;
	int _idx11;
	int _idx12;
	int _idx13;
	int _idx14;
	int _idx15;
	int _idx2;
	int _idx3;
	int _idx4;
	int _idx5;
	int _idx6;
	int _idx7;
	int _idx8;
	int _idx9;
	int chunk_idx_x;
	int chunk_idx_x_max;
	int chunk_idx_y;
	int chunk_idx_y_max;
	int chunk_idx_z;
	int chunk_idx_z_max;
	int idx_1_2;
	int size_1_1;
	int size_1_2;
	//int t;
	int thd_idx_x;
	int thd_idx_y;
	int thd_idx_z;
	int thdblks_idx_x;
	int thdblks_idx_x_max;
	int thdblks_idx_y;
	int thdblks_idx_y_max;
	int thdblks_idx_z;
	int thdblks_idx_z_max;
	int tmp;
	/*
	Initializations
	*/
	size_1_1=(y_max/blockDim.y);
	size_1_2=(z_max/blockDim.z);
	idx_1_2=(blockIdx.y/size_1_2);
	tmp=(blockIdx.y-(idx_1_2*size_1_2));
	chunk_idx_x=(c*(threadIdx.x+(blockDim.x*blockIdx.x)));
	chunk_idx_x_max=(chunk_idx_x+c);
	chunk_idx_y=(threadIdx.y+(tmp*blockDim.y));
	chunk_idx_y_max=(chunk_idx_y+1);
	chunk_idx_z=(threadIdx.z+(idx_1_2*blockDim.z));
	chunk_idx_z_max=(chunk_idx_z+1);
	thdblks_idx_x=(tbx*(threadIdx.x+(blockDim.x*blockIdx.x)));
	thdblks_idx_x_max=(thdblks_idx_x+tbx);
	thdblks_idx_y=(tby*(threadIdx.y+(tmp*blockDim.y)));
	thdblks_idx_y_max=(thdblks_idx_y+tby);
	thdblks_idx_z=(tbz*(threadIdx.z+(idx_1_2*blockDim.z)));
	thdblks_idx_z_max=(thdblks_idx_z+tbz);
	/*
	Implementation
	*/
	/*
	for t = 1..t_max by 1 parallel 1 <level 0> schedule  { ... }
	*/
	//for (t=1; t<=t_max; t+=1)
	{
		/* Index bounds calculations for iterators in thdblks[t=t, s=(tbx, tby, tbz)][0] */
		/* Index bounds calculations for iterators in chunk[t=t, s=(c, 1, 1)][0] */
		/*
		for POINT thd[t=t, s=(1, 1, 1)][0] of size [1, 1, 1] in chunk[t=t, s=(:, :, :)][0] parallel 1 <level 2> schedule default { ... }
		*/
		{
			/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
			thd_idx_z=chunk_idx_z;
			thd_idx_y=chunk_idx_y;
			for (thd_idx_x=chunk_idx_x; thd_idx_x<(chunk_idx_x_max-0); thd_idx_x+=1)
			{
				/* Index bounds calculations for iterators in thd[t=t, s=(1, 1, 1)][0] */
				/*
				u[t=(t+1), s=thd[t=?, s=?][0]][0]=stencil(u[t=t, s=thd[t=?, s=?][0]][0])
				*/
				/* _idx0 = (((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t)) */
				_idx0=(((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t));
				u__u_0[(t-1)][_idx0]=0.1;
				/* _idx1 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+1) */
				_idx1=(_idx0+1);
				u__u_0[(t-1)][_idx1]=0.1;
				/* _idx2 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+2) */
				_idx2=(_idx1+1);
				u__u_0[(t-1)][_idx2]=0.1;
				/* _idx3 = (((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+(((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+3) */
				_idx3=(((_idx1-(3*x_max))-(15*t))+2);
				u__u_0[(t-1)][_idx3]=0.1;
				/* _idx4 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+1)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(5*t))+3) */
				_idx4=((_idx3+x_max)+(5*t));
				u__u_0[(t-1)][_idx4]=0.1;
				/* _idx5 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+2)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(10*t))+3) */
				_idx5=((_idx4+x_max)+(5*t));
				u__u_0[(t-1)][_idx5]=0.1;
				/* _idx6 = (((((((((thd_idx_z*x_max)+((5*t)*thd_idx_z))*y_max)+(((((5*t)*thd_idx_z)+thd_idx_y)+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(15*t))+3) */
				_idx6=((((_idx1+(((-3*x_max)-(15*t))*y_max))-((15*t)*x_max))-(75*(t*t)))+2);
				u__u_0[(t-1)][_idx6]=0.1;
				/* _idx7 = ((((((((((((thd_idx_z+1)*x_max)+((5*t)*thd_idx_z))+(5*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(5*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(25*(t*t)))+(15*t))+3) */
				_idx7=(((_idx6+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u__u_0[(t-1)][_idx7]=0.1;
				/* _idx8 = ((((((((((((thd_idx_z+2)*x_max)+((5*t)*thd_idx_z))+(10*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(10*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(50*(t*t)))+(15*t))+3) */
				_idx8=(((_idx7+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u__u_0[(t-1)][_idx8]=0.1;
				/* _idx9 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+3) */
				_idx9=(_idx1+2);
				u__u_0[(t-1)][_idx9]=0.1;
				/* _idx10 = ((((((((((((thd_idx_z+4)*x_max)+((5*t)*thd_idx_z))+(20*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(20*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(100*(t*t)))+(15*t))+3) */
				_idx10=(((_idx9+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u__u_0[(t-1)][_idx10]=0.1;
				/* _idx11 = ((((((((((((thd_idx_z+5)*x_max)+((5*t)*thd_idx_z))+(25*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(25*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(125*(t*t)))+(15*t))+3) */
				_idx11=(((_idx10+((x_max+(5*t))*y_max))+((5*t)*x_max))+(25*(t*t)));
				u__u_0[(t-1)][_idx11]=0.1;
				/* _idx12 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+4)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(20*t))+3) */
				_idx12=((_idx9+x_max)+(5*t));
				u__u_0[(t-1)][_idx12]=0.1;
				/* _idx13 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+5)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(25*t))+3) */
				_idx13=((_idx12+x_max)+(5*t));
				u__u_0[(t-1)][_idx13]=0.1;
				/* _idx14 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+4) */
				_idx14=(_idx1+3);
				u__u_0[(t-1)][_idx14]=0.1;
				/* _idx15 = ((((((((((((thd_idx_z+3)*x_max)+((5*t)*thd_idx_z))+(15*t))*y_max)+((((((5*t)*thd_idx_z)+thd_idx_y)+(15*t))+3)*x_max))+((25*(t*t))*thd_idx_z))+((5*t)*thd_idx_y))+thd_idx_x)+(75*(t*t)))+(15*t))+5) */
				_idx15=(_idx1+4);
				u__u_0[(t-1)][_idx15]=0.1;
				u__u_0[t][_idx9]=1.1;
			}
		}
	}
}

